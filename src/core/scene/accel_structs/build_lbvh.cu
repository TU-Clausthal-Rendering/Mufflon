#include "hip/hip_runtime.h"
#include "lbvh.hpp"
#include "util/parallel.hpp"
#include "util/types.hpp"
#include "core/cuda/cu_lib_wrapper.hpp"
#include "core/cuda/cuda_utils.hpp"
#include "core/math/sfcurves.hpp"
#include "core/memory/residency.hpp"
#include "core/memory/allocator.hpp"
#include "accel_structs_commons.hpp"

#include <hip/hip_runtime_api.h>
#include <ei/3dtypes.hpp>
#include <>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>




namespace mufflon {
namespace scene {
namespace accel_struct {


static_assert(MAX_ACCEL_STRUCT_PARAMETER_SIZE >= sizeof(LBVH),
			  "Descriptor parameter block to small for this acceleration structure.");


// Sentinel to detect end of parents[] pointer hierarchy
constexpr i32 TreeHead = 0x10000000;


// Type trait to derive some dependent types from a descriptor.
template<typename Desc> struct desc_info {};
template<Device dev> struct desc_info<LodDescriptor<dev>> {
	using MortonCode = u64;
	using PrimCount = ei::IVec3;
	using CostFactor = ei::Vec4;
	static constexpr float NODE_TRAVERSAL_COST = 3.0f;
	static constexpr ei::Vec3 PRIM_TRAVERSAL_COST = { 1.2f, 2.4f, 1.0f };
};
template<Device dev> struct desc_info<SceneDescriptor<dev>> {
	using MortonCode = u64;
	using PrimCount = ei::Vec<i32, 1>;
	using CostFactor = ei::Vec2;
	static constexpr float NODE_TRAVERSAL_COST = 1.0f;
	static constexpr ei::Vec<float, 1> PRIM_TRAVERSAL_COST{ 200000.0f };// TODO: find value for this.
};
template<typename Desc>
using MortonCode_t = typename desc_info<Desc>::MortonCode;
template<typename Desc>
using PrimCount_t = typename desc_info<Desc>::PrimCount;
template<typename Desc>
using CostFactor_t = typename desc_info<Desc>::CostFactor;


// TODO: move to math header ----
// Calculates the point morton code using 63 bits.
template<typename Code>
__forceinline__ __host__ __device__ Code calculate_morton_code(const ei::Vec3& point) {}
template<>
__forceinline__ __host__ __device__ u32 calculate_morton_code<u32>(const ei::Vec3& point) {
	// Discretize the unit cube into a 10 bit integer
	ei::UVec3 discretized{ ei::clamp(point * 1024.0f, 0.0f, 1023.0f) };

	return math::part_by_two10(discretized[0]) * 4
		+ math::part_by_two10(discretized[1]) * 2
		+ math::part_by_two10(discretized[2]);
}
template<>
__forceinline__ __host__ __device__ u64 calculate_morton_code<u64>(const ei::Vec3& point) {
	// Discretize the unit cube into a 21 bit integer
	ei::UVec3 discretized{ ei::clamp(point * 2097152.0f, 0.0f, 2097151.0f) };

	return math::part_by_two21(discretized[0]) * 4
		+ math::part_by_two21(discretized[1]) * 2
		+ math::part_by_two21(discretized[2]);
}
// ----

__forceinline__ __host__ __device__
ei::Vec3 normalize_position(ei::Vec3 pos, const ei::Box& box) {
	ei::Vec3 span = box.max - box.min;
	return sdiv(pos - box.min, span);
}

template<typename DescType>
CUDA_FUNCTION MortonCode_t<DescType>
calculate_morton_code(const DescType& primitives, i32 idx,
					  const ei::Box& sceneBB) {
	const ei::Vec3 centroid = get_centroid(primitives, idx);
	const ei::Vec3 normalizedPos = normalize_position(centroid, sceneBB);
	mAssert(normalizedPos.x >= 0.0f && normalizedPos.x <= 1.0f
		 && normalizedPos.y >= 0.0f && normalizedPos.y <= 1.0f
		 && normalizedPos.z >= 0.0f && normalizedPos.z <= 1.0f);
	return calculate_morton_code<MortonCode_t<DescType>>(normalizedPos);
}


template < typename DescType >
__global__ void calculate_morton_codesD(
	const DescType* desc,
	const ei::Box sceneBB,
	const i32 numPrimitives,
	MortonCode_t<DescType>* mortonCodes,
	i32* sortIndices) {
	const i32 idx = threadIdx.x + blockIdx.x * blockDim.x;

	if(idx >= numPrimitives)
		return;

	mortonCodes[idx] = calculate_morton_code(*desc, idx, sceneBB);
	sortIndices[idx] = idx;
}


// TODO --- cleanup and move to some CUDA header
template<typename T>
void get_maximum_occupancy(i32 &gridSize, i32 &blockSize, i32 totalThreads, T func,
						   i32 dynamicSMemSize = 0) {
	//blockSize;   // The launch configurator returned block size 
	i32 minGridSize; // The minimum grid size needed to achieve the 
	// maximum occupancy for a full device launch 
	//gridSize;    // The actual grid size needed, based on input size 

	cuda::check_error(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, func, dynamicSMemSize, 0));

	if(blockSize != 0)
		// Round up according to array size 
		gridSize = (totalThreads + blockSize - 1) / blockSize;
	else
		gridSize = minGridSize;
}

template<typename UnaryFunction, typename T>
void get_maximum_occupancy_variable_smem(i32 &gridSize, i32 &blockSize, i32 totalThreads, T func,
										 UnaryFunction blockSizeToDynamicSMemSize) {
	//blockSize;   // The launch configurator returned block size 
	i32 minGridSize; // The minimum grid size needed to achieve the 
	// maximum occupancy for a full device launch 
	//gridSize;    // The actual grid size needed, based on input size 

	cuda::check_error(hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, func, blockSizeToDynamicSMemSize, 0));

	if(blockSize != 0)
		// Round up according to array size 
		gridSize = (totalThreads + blockSize - 1) / blockSize;
	else
		gridSize = minGridSize;
}
// -----

template<typename Key>
CUDA_FUNCTION i32 longestCommonPrefix(Key* sortedKeys,
									  i32 numberOfElements, i32 index1, i32 index2, Key key1) {
	// No need to check the upper bound, since i+1 will be at most numberOfElements - 1 (one 
	// thread per internal node)
	if(index2 < 0 || index2 >= numberOfElements)
		return 0;

	Key key2 = sortedKeys[index2];

	if(key1 == key2)
		return 64 + (i32)cuda::clz(u32(index1 ^ index2));

	return (i32)cuda::clz(key1 ^ key2);
}

template <typename T> CUDA_FUNCTION void build_lbvh_tree(
	i32 numPrimitives,
	T* sortedKeys,
	i32 *parents,
	const i32 idx
) {
	const T key1 = sortedKeys[idx];

	const i32 lcp1 = longestCommonPrefix(sortedKeys, numPrimitives, idx, idx + 1, key1);
	const i32 lcp2 = longestCommonPrefix(sortedKeys, numPrimitives, idx, idx - 1, key1);

	const i32 direction = ei::sgn(lcp1 - lcp2);

	// Compute upper bound for the length of the range.
	const i32 minLcp = longestCommonPrefix(sortedKeys, numPrimitives, idx, idx - direction, key1);
	i32 lMax = 128;
	while(longestCommonPrefix(sortedKeys, numPrimitives, idx, idx + lMax * direction, key1) >
		  minLcp) {
		lMax *= 4;
	}

	// Find other end using binary search.
	i32 l = 0;
	i32 t = lMax;
	while(t > 1) {
		t = t / 2;
		if(longestCommonPrefix(sortedKeys, numPrimitives, idx, idx + (l + t) * direction, key1) >
		   minLcp) {
			l += t;
		}
	}
	const i32 j = idx + l * direction;

	// Find the split position using binary search.
	const i32 nodeLcp = longestCommonPrefix(sortedKeys, numPrimitives, idx, j, key1);
	i32 s = 0;
	i32 divisor = 2;
	t = l;
	const i32 maxDivisor = 1 << (32 - cuda::clz(u32(l)));
	while(divisor <= maxDivisor) {
		t = (l + divisor - 1) / divisor;
		if(longestCommonPrefix(sortedKeys, numPrimitives, idx, idx + (s + t) * direction, key1)
			> nodeLcp) {
			s += t;
		}
		divisor *= 2;
	}
	const i32 splitPosition = idx + s * direction + min(direction, 0);

	i32 leftIndex = (min(idx, j) == splitPosition) ?
		splitPosition + numPrimitives - 1
		: splitPosition;
	i32 rightIndex = (max(idx, j) == (splitPosition + 1)) ?
		splitPosition + numPrimitives
		: splitPosition + 1;

	mAssert(leftIndex < 2 * numPrimitives - 1);
	mAssert(rightIndex < 2 * numPrimitives - 1);

	// Set parent nodes.
	parents[leftIndex] = ~idx;
	parents[rightIndex] = idx;

	// Set the parent of the root node to a marker.
	if(idx == 0)
		parents[0] = TreeHead;
}

// Note: dataIndices is of length numPrimitives.
template <typename T> __global__ void build_lbvh_treeD(
	u32 numPrimitives,
	T* sortedKeys,
	i32 *parents
) {
	const i32 idx = threadIdx.x + blockIdx.x * blockDim.x;

	// Check for valid threads.
	if(idx >= (numPrimitives - 1))
		return;

	build_lbvh_tree<T>(numPrimitives, sortedKeys, parents, idx);
}

struct BBCache {
	BBCache() = default;
	CUDA_FUNCTION __forceinline__ BBCache(const ei::Vec4& a, const ei::Vec4& b) :
		boxMin{ a }, primCount{ float_bits_as_int(a.w) },
		boxMax{ b }, cost{ b.w }
	{}
	CUDA_FUNCTION __forceinline__ BBCache(const ei::Vec3& boxMin,
										  float cost,
										  const ei::Vec3& boxMax,
										  i32 primCount) :
		boxMin(boxMin), primCount(primCount), boxMax(boxMax), cost(cost) {}
	ei::Vec3 boxMin;
	i32 primCount;
	ei::Vec3 boxMax;
	float cost;
};
static_assert(sizeof(BBCache) == 8 * sizeof(float), "Alignment of BBCache is broken.");

struct BoundingBoxFunctor {
	__host__ __device__
		i32 operator()(i32 i) const { return sizeof(BBCache) * i; }
};


template < typename DescType > CUDA_FUNCTION
PrimCount_t<DescType> get_count(const DescType&, i32 primIdx) { return PrimCount_t<DescType>{1}; }
template < Device dev > CUDA_FUNCTION
PrimCount_t<LodDescriptor<dev>> get_count(const LodDescriptor<dev>& obj, i32 primIdx) {
	if(primIdx >= i32(obj.polygon.numTriangles + obj.polygon.numQuads))
		return { 0, 0, 1 };
	if(primIdx >= i32(obj.polygon.numTriangles))
		return { 0, 1, 0 };
	return { 1, 0, 0 };
}

template< typename PrimCount > CUDA_FUNCTION
i32 encode_prim_counts(const PrimCount& primCount) {
	return primCount.x;
}
template<> CUDA_FUNCTION
i32 encode_prim_counts(const ei::IVec3& primCount) {
	return (ei::min(primCount.x, 0x3FF) << 20)
		| (ei::min(primCount.y, 0x3FF) << 10)
		| (ei::min(primCount.z, 0x3FF));
}

template< typename PrimCount > CUDA_FUNCTION
PrimCount extract_prim_counts(i32 primCount) {
	return PrimCount{ primCount };
}
template<> CUDA_FUNCTION
ei::IVec3 extract_prim_counts(i32 primCount) {
	return ei::IVec3{ (primCount & 0x3FF00000) >> 20,
					 (primCount & 0x000FFC00) >> 10,
					 (primCount & 0x000003FF) };
}

template < typename DescType >
CUDA_FUNCTION float get_cost(const PrimCount_t<DescType>& primCount) {
	const auto traversalCost = desc_info<DescType>::PRIM_TRAVERSAL_COST;
	return desc_info<DescType>::NODE_TRAVERSAL_COST
		+ dot(primCount, traversalCost);
}

template < typename DescType >
CUDA_FUNCTION void calculate_bounding_boxes(
	const DescType& desc,
	i32 idx,		// Global thread index
	i32 primIdx,	// Index of the primitive for which this thread is responsible
	i32 numPrimitives,
	const i32* __restrict__ parents,
	ei::Vec4* boundingBoxes,
	cuda::Atomic<DescType::DEVICE, i32> * __restrict__ counters,
	i32* __restrict__ nodeMarks
) {

	ei::Box currentBb = get_bounding_box(desc, primIdx);
	float currentSA = ei::surface(currentBb);

	// Get primitive count of the node and its cost
	auto primitiveCount = get_count(desc, primIdx);
	float cost = get_cost<DescType>(primitiveCount);

	// Store in global array
	const i32 numInternalNodes = numPrimitives - 1;
	const i32 leafIndex = idx + numInternalNodes;
	const i32 boxId = leafIndex << 1;
	boundingBoxes[boxId] = { currentBb.min, int_bits_as_float(encode_prim_counts(primitiveCount)) };
	boundingBoxes[boxId + 1] = { currentBb.max, cost };

	// Proceed upwards in the hierarchy
	i32 currentNode = parents[leafIndex];
	i32 lastNode = idx; // Only for the first iteration is is the thread index, later it is realy a node
	bool lastIsLeftChild = currentNode < 0;
	if(currentNode < 0) currentNode = ~currentNode;

	// In the counters array, we have stored the id of the thread that processed the other
	// children of this node.
	i32 otherChildThreadIdx = cuda::atomic_exchange<DescType::DEVICE>(counters[currentNode], leafIndex);

	// The first thread to reach a node will just die.
	// This circumvents the global sync problem. The second thread
	// can be sure that the data of the first one is present.
	while(otherChildThreadIdx != 0xFFFFFFFF) {
		cuda::globalMemoryBarrier();		// For reads on boundingBoxes[]

		i32 otherChildNode = lastIsLeftChild ? lastNode + 1 : lastNode - 1;
		// otherChildNode may be invalid if it was a leaf - this is encoded
		// in the otherChildThreadIdx.
		if(otherChildThreadIdx >= numInternalNodes) {
			otherChildThreadIdx -= numInternalNodes;	// Remove the encoding, such that it is an index
			otherChildNode += numInternalNodes;			// Get the correct other node
		}

		BBCache childInfo;
		// The children were processed in different blocks, so we have to find out if the one
		// that was not processed by this thread was the left or right one.
		{
			i32 boxId = otherChildNode * 2;
			childInfo = BBCache{ boundingBoxes[boxId], boundingBoxes[boxId + 1] };
		}

		// Compute data for this node
		currentBb.min = ei::min(currentBb.min, childInfo.boxMin);
		currentBb.max = ei::max(currentBb.max, childInfo.boxMax);
		float newSA = ei::surface(currentBb);
		float otherSA = ei::surface(ei::Box{ childInfo.boxMin, childInfo.boxMax });
		auto primitiveCountOther = extract_prim_counts<PrimCount_t<DescType>>(childInfo.primCount);
		cost = desc_info<DescType>::NODE_TRAVERSAL_COST
			+ currentSA / newSA * get_cost<DescType>(primitiveCount)
			+ otherSA / newSA * get_cost<DescType>(primitiveCountOther);
		primitiveCount += primitiveCountOther;
		currentSA = newSA;
		i32 boxId = currentNode << 1;
		boundingBoxes[boxId] = { currentBb.min, int_bits_as_float(encode_prim_counts(primitiveCount)) };
		boundingBoxes[boxId + 1] = { currentBb.max, cost };

		// Go to next node
		lastNode = currentNode;
		currentNode = parents[currentNode];
		lastIsLeftChild = currentNode < 0;
		if(currentNode < 0) currentNode = ~currentNode;
		if(currentNode == TreeHead) break;
		otherChildThreadIdx = cuda::atomic_exchange<DescType::DEVICE>(counters[currentNode], idx);
	}

	// Initialize nodeMarks for the next kernel.
	if(idx < numPrimitives - 1) // Only for internal nodes
		nodeMarks[idx] = 1;
}

// TODO: epsilon scalar overloads
CUDA_FUNCTION __forceinline__ bool greatereq(i32 a, i32 b) { return a >= b; }
CUDA_FUNCTION __forceinline__ bool any(bool a) { return a; }

template< typename DescType >
CUDA_FUNCTION void mark_collapsed_nodes(
	const ei::Vec4* __restrict__ boundingBoxes,
	const i32* __restrict__ parents,
	const i32 leafIndex,	// Global thread index + numInternalNodes
	i32* nodeMarks			// Initialized to 1 (do not delete this node)
) {
	// 1. Pass: Bottom-Up traversal as long as nodes are canditates for collapses.
	// Goal: find the highest collapsable node (ancestor) of the current leaf.
	i32 collapseNode = -1;
	i32 currentNode = parents[leafIndex];
	if(currentNode < 0) currentNode = ~currentNode;
	while(currentNode != 0) {
		ei::Vec4 boxMin_primCount = boundingBoxes[currentNode * 2];
		ei::Vec4 boxMax_primCost = boundingBoxes[currentNode * 2 + 1];
		auto primitiveCount = extract_prim_counts<PrimCount_t<DescType>>(float_bits_as_int(boxMin_primCount.w));

		// Termination condition: more than 1023 primitves of a single type.
		// This is the largest possible value using our encoding.
		if(any(greatereq(primitiveCount, 1023)))
			break;

		float cost = boxMax_primCost.w;		// Cost without collapse
		float costAsLeaf = get_cost<DescType>(primitiveCount);
		if(costAsLeaf < cost)
			collapseNode = currentNode;

		currentNode = parents[currentNode];
		if(currentNode < 0) currentNode = ~currentNode;
	}

	// We cannot set marks imediatelly, since there is counter-example for that
	// SAH guarentees to be monotony.

	// Is there any ancestor, which is collapsed?
	if(collapseNode != -1) {
		// Mark the current node and all nodes up to the ancestor as deleted.
		i32 currentNode = parents[leafIndex];
		if(currentNode < 0) currentNode = ~currentNode;
		while(currentNode != collapseNode) {
			nodeMarks[currentNode] = 0;
			currentNode = parents[currentNode];
			if(currentNode < 0) currentNode = ~currentNode;
		}
		nodeMarks[currentNode] = 0; // Include a mark for the collapsed node
	}
}


template< typename DescType >
__global__ void calculate_bounding_boxesD(
	const DescType* desc,
	const i32 numPrimitives,
	const i32* __restrict__ sortedIndices,
	const i32* __restrict__ parents,
	ei::Vec4* boundingBoxes,
	cuda::Atomic<DescType::DEVICE, i32>* __restrict__ counters,
	i32* __restrict__ nodeMarks
) {
	const i32 idx = threadIdx.x + blockIdx.x * blockDim.x;

	// Check for valid threads.
	if(idx >= numPrimitives)
		return;

	calculate_bounding_boxes<DescType>(*desc, idx, sortedIndices[idx],
									   numPrimitives, parents, boundingBoxes, counters, nodeMarks);
}

template< typename DescType >
__global__ void mark_nodesD(
	u32 numInternalNodes,
	const ei::Vec4* __restrict__ boundingBoxes,
	const i32* __restrict__ parents,
	i32* __restrict__ nodeMarks
) {
	const i32 idx = threadIdx.x + blockIdx.x * blockDim.x;

	if(idx > numInternalNodes) // idx == numInternalNodes OK, because we want one thread per leaf
		return;

	mark_collapsed_nodes<DescType>(boundingBoxes, parents,
								   idx + numInternalNodes, nodeMarks);
}


CUDA_FUNCTION bool is_collapsed(const i32* offsets, const i32 numInternalNodes, const i32 node) {
	bool isCollapsed = false;
	if(node > 0 && node < numInternalNodes) // Root and leaf nodes cannot be collapsed.
		// Revert the inclusive scan to check wether this node had a mark '1' or '0'
		isCollapsed = (offsets[node] - offsets[node - 1]) == 0; // 0 means collapse
	return isCollapsed;
}

// Called for all nodes
template < typename DescType >
CUDA_FUNCTION void copy_to_collapsed_bvh(
	const ei::Vec4* __restrict__ boundingBoxes,
	const i32* __restrict__ parents,
	const i32* __restrict__ offsets,
	const i32 node,
	const i32 numInternalNodes,
	const i32 numInternalNodesAfterCollapse,
	BvhNode* __restrict__ collapsedBVH
) {
	// Collapsed nodes do not exist anymore and do not write anything in the
	// hierarchy
	if(!is_collapsed(offsets, numInternalNodes, node)) {
		// Determine if it is a left or right child. A child will
		// fill half of the data of the parent. To determine which
		// half is written we use 'offset'.
		i32 parent = parents[node];
		i32 offset;
		if(parent < 0) { // Left child?
			offset = 0;
			parent = ~parent;
		} else offset = 1;

		// The parent could be collapsed if the current node is a leaf.
		// Search the first non-collapsed parent.
		i32 finalNode = node;
		if(finalNode >= numInternalNodes) {
			while(is_collapsed(offsets, numInternalNodes, parent)) {
				if(offset == 1) return; // Only the left-most child of the subtree may write the parent node.
				finalNode = parent;
				parent = parents[parent];
				if(parent < 0) { // Left child?
					offset = 0;
					parent = ~parent;
				} else offset = 1;
			}
		}

		// Read the data of the current node (the highest collapsed or the leaf)
		const ei::Vec4 boxMin_primCount = boundingBoxes[finalNode * 2];
		const ei::Vec4 boxMax_primCost = boundingBoxes[finalNode * 2 + 1];
		const i32 countCode = float_bits_as_int(boxMin_primCount.w);
		const i32 primCount = ei::sum(extract_prim_counts<PrimCount_t<DescType>>(countCode));

		const i32 outIdx = (offsets[parent] - 1) * 2;
		const i32 outNode = (node >= numInternalNodes) ?
			node - numInternalNodes + numInternalNodesAfterCollapse // Leaf. Offset address by number of collapsed nodes.
			: offsets[node] - 1;	// Internal node. Get the new position from offset array.
		// Enlarge the bounding box to avoid numerical issues in the tracing
		const ei::Vec3 center = (ei::Vec3{ boxMin_primCount } +ei::Vec3{ boxMax_primCost }) * 0.5f;
		const ei::Vec3 bbMin = (ei::Vec3{ boxMin_primCount } -center) * 1.0001f + center;
		const ei::Vec3 bbMax = (ei::Vec3{ boxMax_primCost } -center) * 1.0001f + center;
		collapsedBVH[outIdx + offset] = { {bbMin, bbMax}, outNode, primCount };
	}
}

template < typename DescType >
__global__ void copy_to_collapsed_bvhD(
	const i32 numNodes,
	const i32 numInternalNodes,
	const i32 numInternalNodesAfterCollapse,
	const ei::Vec4 * __restrict__ boundingBoxes,
	const i32* __restrict__ parents,
	const i32* __restrict__ offsets,
	BvhNode* __restrict__ collapsedBVH
) {
	i32 idx = threadIdx.x + blockIdx.x * blockDim.x + 1;

	if(idx >= numNodes || idx == 0)
		return;

	copy_to_collapsed_bvh<DescType>(boundingBoxes, parents, offsets, idx, numInternalNodes, numInternalNodesAfterCollapse, collapsedBVH);
}


template < typename DescType >
void LBVHBuilder::build_lbvh(const DescType& desc,
							 const ei::Box& currentBB,
							 const i32 numPrimitives
) {
	if(numPrimitives == 1) { // Not necessary to build anything - trace code will skip the BVH
		m_primIds.resize(4); // Make sure there is some memory (needs_rebuild depends on that) TODO: store simple bool instead?
		m_bvhNodes.resize(1);
		mem_set<DescType::DEVICE>(m_primIds.acquire<DescType::DEVICE>(), 0, sizeof(u32));
		m_bvhNodes.acquire<DescType::DEVICE>();
		// It is indeed necessary to have a zero here in case of 1 instance and 1 primitive only
		return;
	}

	const i32 numInternalNodes = numPrimitives - 1;
	const i32 numNodes = numInternalNodes + numPrimitives;

	// Device copy of descriptor
	unique_device_ptr<DescType::DEVICE, DescType> deviceDesc;
	if(DescType::DEVICE != Device::CPU) {
		deviceDesc = make_udevptr<DescType::DEVICE, DescType>();
		copy(deviceDesc.get(), &desc, sizeof(DescType));
	}

	// Allocate memory for a part of the BVH.We do not know the final size yet and
	// cannot allocate the other parts in bvh.
	m_primIds.resize(numPrimitives * sizeof(i32));
	i32* primIds = as<i32>(m_primIds.acquire<DescType::DEVICE>());
	auto parents = make_udevptr_array<DescType::DEVICE, i32, false>(numNodes);

	// To avoid unnecessary allocations we allocate the device counter array here already (usage in calculate_bounding_boxes)
	// The counters buffer is used with atomics to detect the order of executions within the launch.
	unique_device_ptr<DescType::DEVICE, cuda::Atomic<DescType::DEVICE, i32>[]> deviceCounters = nullptr;
	// Temporary buffer used for all kinds of things on CUDA side (nothing on CPU side)
	unique_device_ptr<DescType::DEVICE, u8[]> tmpMem = nullptr;

	const auto maxTmpMemSize = ei::max(
		numPrimitives * sizeof(MortonCode_t<DescType>) // For unsorted morton codes and
		+ numPrimitives * sizeof(i32),					 // primIds at the same time
		numInternalNodes * sizeof(i32)				 // OR deviceCountes later
	);
	if(DescType::DEVICE == Device::CPU) {
		deviceCounters = make_udevptr_array<DescType::DEVICE, cuda::Atomic<DescType::DEVICE, i32>>(maxTmpMemSize / sizeof(i32));
	} else {
		// Allocate a block of temporary memory for several build purposes
		// This is only necessary for CUDA as of now
		tmpMem = make_udevptr_array<DescType::DEVICE, u8, false>(maxTmpMemSize);

		// The device counters may use the temporary memory, but don't get to own it; the pointer is released in calculate_bounding_boxes
		deviceCounters.reset(as<cuda::Atomic<DescType::DEVICE, i32>>(tmpMem.get()));
	}

	// Calculate Morton codes.
	{
		auto sortedMortonCodes = make_udevptr_array<DescType::DEVICE, MortonCode_t<DescType>, false>(numPrimitives);
		if(DescType::DEVICE == Device::CUDA) {
			auto* mortonCodes = as<MortonCode_t<DescType>>(tmpMem.get());
			auto* primIdsUnsorted = as<i32>(tmpMem.get() + numPrimitives * sizeof(MortonCode_t<DescType>));

			i32 numBlocks, numThreads;
			get_maximum_occupancy(numBlocks, numThreads, numPrimitives, calculate_morton_codesD<DescType>);
			calculate_morton_codesD <<< numBlocks, numThreads >>> (
				deviceDesc.get(), currentBB, numPrimitives, mortonCodes, primIdsUnsorted);
			cuda::check_error(hipGetLastError());

			// Sort based on Morton codes.
			CuLib::DeviceSort(numPrimitives, mortonCodes, sortedMortonCodes.get(),
							  primIdsUnsorted, primIds);
			cuda::check_error(hipGetLastError());
		} else {
#pragma PARALLEL_FOR
			for(i32 idx = 0; idx < numPrimitives; idx++) {
				sortedMortonCodes[idx] = calculate_morton_code<DescType>(desc, idx, currentBB);
				primIds[idx] = idx;
			}

			// Sort based on Morton codes.
			thrust::sort_by_key(sortedMortonCodes.get(), sortedMortonCodes.get() + numPrimitives, primIds);
		}

		// Create BVH.
		// Layout: first internal nodes, then leves.
		if(DescType::DEVICE == Device::CUDA) {
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(build_lbvh_treeD<MortonCode_t<DescType>>), hipFuncCachePreferL1);
			i32 numBlocks, numThreads;
			get_maximum_occupancy(numBlocks, numThreads, numInternalNodes, build_lbvh_treeD<MortonCode_t<DescType>>);
			build_lbvh_treeD<MortonCode_t<DescType>> <<< numBlocks, numThreads >>> (
				numPrimitives,
				sortedMortonCodes.get(),
				parents.get());
			cuda::check_error(hipGetLastError());
		} else {
#pragma PARALLEL_FOR
			for(i32 idx = 0; idx < numInternalNodes; idx++)
				build_lbvh_tree<MortonCode_t<DescType>>(numPrimitives,
														sortedMortonCodes.get(), parents.get(), idx);
		}
	}

	// Calculate bounding boxes and SAH.
	auto boundingBoxes = make_udevptr_array<DescType::DEVICE, ei::Vec4, false>(numNodes * 2);
	mem_set<DescType::DEVICE>(deviceCounters.get(), 0xFF, numInternalNodes * sizeof(i32));
	// Allocate some memory of the later computation of partial BVH collapses.
	// This memory is initialized in the calculate_bounding_boxesD kernel to 1,
	// because setting 32bit integers to 1 is not possible with a memSet.
	auto collapseOffsets = make_udevptr_array<DescType::DEVICE, i32, false>(numInternalNodes);
	if(DescType::DEVICE == Device::CUDA) {
		// Calculate BVH bounding boxes.
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(calculate_bounding_boxesD<DescType>), hipFuncCachePreferShared);
		BoundingBoxFunctor functor;
		i32 numBlocks, numThreads;
		get_maximum_occupancy_variable_smem(numBlocks, numThreads, numPrimitives,
											calculate_bounding_boxesD<DescType>, functor);
		const i32 bboxCacheSize = numThreads * sizeof(BBCache);
		calculate_bounding_boxesD <<< numBlocks, numThreads, bboxCacheSize >>> (
			deviceDesc.get(), numPrimitives, primIds, parents.get(),
			boundingBoxes.get(), deviceCounters.get(),
			collapseOffsets.get()
			);
		cuda::check_error(hipGetLastError());

		// We need to release the pointer from deviceCounters here since the deallocation is part of tmpMem for CUDA side
		deviceCounters.release();
	} else {
#pragma PARALLEL_FOR
		for(i32 idx = 0; idx < numPrimitives; idx++) {
			calculate_bounding_boxes(desc, idx, primIds[idx],
									 numPrimitives, parents.get(), boundingBoxes.get(),
									 deviceCounters.get(), collapseOffsets.get());
		}
	}

	// Find out which nodes can be collapsed according to SAH.
	if(DescType::DEVICE == Device::CUDA) {
		i32 numBlocks, numThreads;
		get_maximum_occupancy(numBlocks, numThreads, numPrimitives, mark_nodesD<DescType>);
		mark_nodesD<DescType> <<< numBlocks, numThreads >>> (numInternalNodes,
															   boundingBoxes.get(), parents.get(),
															   collapseOffsets.get()
															   );
		cuda::check_error(hipGetLastError());
	} else {
#pragma PARALLEL_FOR
		for(i32 idx = 0; idx < numPrimitives; idx++) {
			mark_collapsed_nodes<DescType>(boundingBoxes.get(), parents.get(),
										   idx + numInternalNodes, collapseOffsets.get());
		}
	}

	// Scan to get values for offsets.
	i32 numRemovedInternalNodes;
	if(DescType::DEVICE == Device::CUDA) {
		CuLib::DeviceInclusiveSum(numInternalNodes, collapseOffsets.get(), collapseOffsets.get());
		copy(&numRemovedInternalNodes, collapseOffsets.get() + numInternalNodes - 1, sizeof(i32));
		numRemovedInternalNodes = numInternalNodes - numRemovedInternalNodes;
	} else {
		// Scan to get values for offsets.
		thrust::inclusive_scan(collapseOffsets.get(), collapseOffsets.get() + numInternalNodes, collapseOffsets.get());
		numRemovedInternalNodes = numInternalNodes - collapseOffsets[numInternalNodes - 1];
	}

	logPedantic("[LBVHBuilder::build_lbvh] collapsing removed ", numRemovedInternalNodes, " nodes.");

	// Write the final compacted BVH
	i32 numNodesInCollapsedBVH = (numInternalNodes - numRemovedInternalNodes);
	m_bvhNodes.resize(numNodesInCollapsedBVH * sizeof(ei::Vec4) * 4);
	BvhNode* collapsedBVH = as<BvhNode>(m_bvhNodes.acquire<DescType::DEVICE>());
	if(DescType::DEVICE == Device::CUDA) {
		i32 numBlocks, numThreads;
		get_maximum_occupancy(numBlocks, numThreads, numNodes, copy_to_collapsed_bvhD<DescType>);
		copy_to_collapsed_bvhD<DescType> <<< numBlocks, numThreads >>> (
			numNodes, numInternalNodes, numNodesInCollapsedBVH, boundingBoxes.get(), parents.get(),
			collapseOffsets.get(), collapsedBVH);
		cuda::check_error(hipGetLastError());
	} else {
#pragma PARALLEL_FOR
		for(i32 idx = 1; idx < numNodes; ++idx)
			copy_to_collapsed_bvh<DescType>(
				boundingBoxes.get(), parents.get(), collapseOffsets.get(),
				idx, numInternalNodes, numNodesInCollapsedBVH, collapsedBVH);
	}
}

template < Device dev >
void LBVHBuilder::build(LodDescriptor<dev>& obj, const ei::Box& currentBB) {
	build_lbvh<LodDescriptor<dev>>(obj, currentBB, obj.numPrimitives);
	m_primIds.mark_changed(dev);
	m_bvhNodes.mark_changed(dev);
}

template void LBVHBuilder::build<Device::CPU>(LodDescriptor<Device::CPU>&, const ei::Box&);
template void LBVHBuilder::build<Device::CUDA>(LodDescriptor<Device::CUDA>&, const ei::Box&);

template < Device dev >
void LBVHBuilder::build(
	const SceneDescriptor<dev>& scene
) {
	logInfo("[LBVHBuilder::build] Building BVH for ", scene.numInstances, " instances.");
	build_lbvh<SceneDescriptor<dev>>(scene, scene.aabb, scene.numInstances);
	m_primIds.mark_changed(dev);
	m_bvhNodes.mark_changed(dev);
}

template void LBVHBuilder::build<Device::CPU>(const SceneDescriptor<Device::CPU>&);
template void LBVHBuilder::build<Device::CUDA>(const SceneDescriptor<Device::CUDA>&);

}
}
} // namespace mufflon
