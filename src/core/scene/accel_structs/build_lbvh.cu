#include "hip/hip_runtime.h"
#include "lbvh.hpp"
#include "util/types.hpp"
#include "core/cuda/cu_lib_wrapper.h"
#include "core/math/sfcurves.hpp"
#include "core/memory/residency.hpp"
#include "core/memory/allocator.hpp"
#include "accel_structs_commons.hpp"

#include <hip/hip_runtime_api.h>
#include <ei/3dtypes.hpp>
#include <>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#ifdef _MSC_VER
#include <intrin.h>
#endif // _MSC_VER


//namespace mufflon { namespace {//TODO use anonym?
namespace mufflon {
namespace scene {
namespace accel_struct {


static_assert(MAX_ACCEL_STRUCT_PARAMETER_SIZE >= sizeof(LBVH),
	"Descriptor parameter block to small for this acceleration structure.");

CUDA_FUNCTION void syncthreads() {
#ifdef __CUDA_ARCH__
	__syncthreads();
#endif // __CUDA_ARCH__
}

CUDA_FUNCTION u64 clzll(u64 v) {
#ifdef __CUDA_ARCH__
	return __clzll(v);
#else
#ifdef _MSC_VER
	return __lzcnt64(v);
#else
	return (v == 0) ? 64 : 63 - (u64)log2f((float)v);
#endif // _MSC_VER
#endif // __CUDA_ARCH__
}

CUDA_FUNCTION u32 clz(u32 v) {
#ifdef __CUDA_ARCH__
	return __clz(v);
#else
#ifdef _MSC_VER
	return __lzcnt(v);
#else
	return (v == 0) ? 32 : 31 - (u32)log2f((float)v);
#endif // _MSC_VER
#endif // __CUDA_ARCH__
}


// Generic centroid overloads.
// This helps in generalizing the code of a builder
template < Device dev >
__host__ __device__ ei::Vec3 get_centroid(const ObjectDescriptor<dev>& obj, i32 idx) {
	// Primitve order: Trianges, Quads, Spheres -> idx determines the case
	i32 spheresOffset = obj.polygon.numQuads + obj.polygon.numTriangles;
	if(idx >= spheresOffset)
		return obj.spheres.spheres[idx - spheresOffset].center;
	if(idx >= i32(obj.polygon.numTriangles)) {
		i32 quadId = (idx - obj.polygon.numTriangles) << 2;
		return (obj.polygon.vertices[obj.polygon.vertexIndices[quadId  ]]
			  + obj.polygon.vertices[obj.polygon.vertexIndices[quadId+1]]
			  + obj.polygon.vertices[obj.polygon.vertexIndices[quadId+2]]
			  + obj.polygon.vertices[obj.polygon.vertexIndices[quadId+3]]) / 4.0f;
	}
	i32 triId = idx * 3;
	return (obj.polygon.vertices[obj.polygon.vertexIndices[triId  ]]
		  + obj.polygon.vertices[obj.polygon.vertexIndices[triId+1]]
		  + obj.polygon.vertices[obj.polygon.vertexIndices[triId+2]]) / 3.0f;
}

template < Device dev >
__host__ __device__ ei::Vec3 get_centroid(const SceneDescriptor<dev>& scene, i32 idx) {
	i32 objIdx = scene.objectIndices[idx];
	//const ei::Box aabb = ei::transform(prim.objAabbs[objIdx], prim.matrices[idx]);
	// Extract the translation from the matrix only (no need to compute the
	// full bounding box.
	return center(scene.aabbs[objIdx]) + ei::Vec3{scene.transformations[idx][3],
												  scene.transformations[idx][7],
												  scene.transformations[idx][11]};
}

// Generic bounding box overloads.
// This helps in generalizing the code of a builder
template < Device dev >
__host__ __device__ ei::Box get_bounding_box(const ObjectDescriptor<dev>& obj, i32 idx) {
	// Primitve order: Trianges, Quads, Spheres -> idx determines the case
	i32 spheresOffset = obj.polygon.numQuads + obj.polygon.numTriangles;
	if(idx >= spheresOffset)
		return ei::Box(obj.spheres.spheres[idx - spheresOffset]);
	if(idx >= i32(obj.polygon.numTriangles)) {
		i32 quadId = (idx - obj.polygon.numTriangles) << 2;
		return ei::Box(obj.polygon.vertices[obj.polygon.vertexIndices[quadId  ]],
					   obj.polygon.vertices[obj.polygon.vertexIndices[quadId+1]],
					   obj.polygon.vertices[obj.polygon.vertexIndices[quadId+2]],
					   obj.polygon.vertices[obj.polygon.vertexIndices[quadId+3]]);
	}
	i32 triId = idx * 3;
	return ei::Box(obj.polygon.vertices[obj.polygon.vertexIndices[triId  ]],
				   obj.polygon.vertices[obj.polygon.vertexIndices[triId+1]],
				   obj.polygon.vertices[obj.polygon.vertexIndices[triId+2]]);
}

template < Device dev >
__host__ __device__ ei::Box get_bounding_box(const SceneDescriptor<dev>& scene, i32 idx) {
	i32 objIdx = scene.objectIndices[idx];
	return ei::transform(scene.aabbs[objIdx], scene.transformations[idx]);
}


// Calculates the point morton code using 63 bits.
template<typename Code>
__forceinline__ __host__ __device__ Code calculate_morton_code(const ei::Vec3& point) {}
template<>
__forceinline__ __host__ __device__ u32 calculate_morton_code<u32>(const ei::Vec3& point) {
	// Discretize the unit cube into a 10 bit integer
	ei::UVec3 discretized{ ei::clamp(point * 1024.0f, 0.0f, 1023.0f) };

	return math::part_by_two10(discretized[0]) * 4
		+ math::part_by_two10(discretized[1]) * 2
		+ math::part_by_two10(discretized[2]);
}
template<>
__forceinline__ __host__ __device__ u64 calculate_morton_code<u64>(const ei::Vec3& point) {
	// Discretize the unit cube into a 21 bit integer
	ei::UVec3 discretized { ei::clamp(point * 2097152.0f, 0.0f, 2097151.0f) };

	return math::part_by_two21(discretized[0]) * 4
		 + math::part_by_two21(discretized[1]) * 2
		 + math::part_by_two21(discretized[2]);
}

__forceinline__ __host__ __device__
ei::Vec3 normalize_position(ei::Vec3 pos, const ei::Box& box) {
	ei::Vec3 span = box.max - box.min;
	return (pos - box.min) / span;
}

template<typename DescType, typename Code>
CUDA_FUNCTION Code calculate_morton_code(const DescType& primitives, i32 idx,
										 const ei::Box& sceneBB) {
	const ei::Vec3 centroid = get_centroid(primitives, idx);
	const ei::Vec3 normalizedPos = normalize_position(centroid, sceneBB);
	return calculate_morton_code<Code>(normalizedPos);
}


__global__ void calculate_morton_codes64D(
	const ObjectDescriptor<Device::CUDA>& obj,
	const ei::Box& sceneBB,
	u64* mortonCodes,
	i32* sortIndices) {
	const i32 idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx >= obj.numPrimitives)
		return;

	mortonCodes[idx] = calculate_morton_code<ObjectDescriptor<Device::CUDA>, u64>(obj, idx, sceneBB);
	sortIndices[idx] = idx;
}

__global__ void calculate_morton_codes32D(
	const SceneDescriptor<Device::CUDA>& scene,
	const ei::Box& sceneBB,
	u32* mortonCodes,
	i32* sortIndices) {
	const i32 idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx >= scene.numInstances)
		return;

	mortonCodes[idx] = calculate_morton_code<SceneDescriptor<Device::CUDA>, u32>(scene, idx, sceneBB);
	sortIndices[idx] = idx;
}

template<typename T> 
void get_maximum_occupancy(i32 &gridSize, i32 &blockSize, i32 totalThreads, T func, 
						   i32 dynamicSMemSize = 0)
{
	//blockSize;   // The launch configurator returned block size 
	i32 minGridSize; // The minimum grid size needed to achieve the 
	// maximum occupancy for a full device launch 
	//gridSize;    // The actual grid size needed, based on input size 

	cuda::check_error(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, func, dynamicSMemSize, 0));

	if (blockSize != 0)
		// Round up according to array size 
		gridSize = (totalThreads + blockSize - 1) / blockSize;
	else
		gridSize = minGridSize;
}

struct BoundingBoxFunctor {
	__host__ __device__
	i32 operator()(i32 i) const { return sizeof(ei::Vec4) * 2 * i; }
};

template<typename UnaryFunction, typename T>
void get_maximum_occupancy_variable_smem(i32 &gridSize, i32 &blockSize, i32 totalThreads, T func,
	UnaryFunction blockSizeToDynamicSMemSize)
{
	//blockSize;   // The launch configurator returned block size 
	i32 minGridSize; // The minimum grid size needed to achieve the 
	// maximum occupancy for a full device launch 
	//gridSize;    // The actual grid size needed, based on input size 

	cuda::check_error(hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, func, blockSizeToDynamicSMemSize, 0));

	if (blockSize != 0)
		// Round up according to array size 
		gridSize = (totalThreads + blockSize - 1) / blockSize;
	else
		gridSize = minGridSize;
}

CUDA_FUNCTION i32 longestCommonPrefix(u64* sortedKeys,
	i32 numberOfElements, i32 index1, i32 index2, u64 key1)
{
	// No need to check the upper bound, since i+1 will be at most numberOfElements - 1 (one 
	// thread per internal node)
	if (index2 < 0 || index2 >= numberOfElements)
	{
		return 0;
	}

	u64 key2 = sortedKeys[index2];

	if (key1 == key2)
	{
		return 64 + (i32)clzll(index1 ^ index2);
	}

	return (i32)clzll(key1 ^ key2);
}

CUDA_FUNCTION i32 longestCommonPrefix(u32* sortedKeys,
	i32 numberOfElements, i32 index1, i32 index2, u32 key1)
{
	// No need to check the upper bound, since i+1 will be at most numberOfElements - 1 (one 
	// thread per internal node)
	if (index2 < 0 || index2 >= numberOfElements)
	{
		return 0;
	}

	u32 key2 = sortedKeys[index2];

	if (key1 == key2)
	{
		return 32 + clz(index1 ^ index2);
	}

	return clz(key1 ^ key2);
}

CUDA_FUNCTION i32 sgn(i32 number)
{
	return (0 < number) - (0 > number);
}

template <typename T> CUDA_FUNCTION void build_lbvh_tree(
	u32 numPrimitives,
	T* sortedKeys,
	i32 *parents,
	const i32 idx
) {
	const T key1 = sortedKeys[idx];

	const i32 lcp1 = longestCommonPrefix(sortedKeys, numPrimitives, idx, idx + 1, key1);
	const i32 lcp2 = longestCommonPrefix(sortedKeys, numPrimitives, idx, idx - 1, key1);

	const i32 direction = sgn((lcp1 - lcp2));

	// Compute upper bound for the length of the range.
	const i32 minLcp = longestCommonPrefix(sortedKeys, numPrimitives, idx, idx - direction, key1);
	i32 lMax = 128;
	while (longestCommonPrefix(sortedKeys, numPrimitives, idx, idx + lMax * direction, key1) >
		minLcp)
	{
		lMax *= 4;
	}

	// Find other end using binary search.
	i32 l = 0;
	i32 t = lMax;
	while (t > 1)
	{
		t = t / 2;
		if (longestCommonPrefix(sortedKeys, numPrimitives, idx, idx + (l + t) * direction, key1) >
			minLcp)
		{
			l += t;
		}
	}
	const i32 j = idx + l * direction;

	// Find the split position using binary search.
	const i32 nodeLcp = longestCommonPrefix(sortedKeys, numPrimitives, idx, j, key1);
	i32 s = 0;
	i32 divisor = 2;
	t = l;
	const i32 maxDivisor = 1 << (32 - clz(l));
	while (divisor <= maxDivisor)
	{
		t = (l + divisor - 1) / divisor;
		if (longestCommonPrefix(sortedKeys, numPrimitives, idx, idx + (s + t) * direction, key1) >
			nodeLcp)
		{
			s += t;
		}
		divisor *= 2;
	}
	const i32 splitPosition = idx + s * direction + min(direction, 0);

	i32 leftIndex;
	i32 rightIndex;

	// Update left child pointer to a leaf.
	if (min(idx, j) == splitPosition)
	{
		// Children is a leaf, add the number of internal nodes to the index.
		leftIndex = splitPosition + numPrimitives - 1;
	}
	else
	{
		leftIndex = splitPosition;
	}

	// Update right child pointer to a leaf.
	if (max(idx, j) == (splitPosition + 1))
	{
		// Children is a leaf, add the number of internal nodes to the index.
		rightIndex = splitPosition + numPrimitives;
	}
	else
	{
		rightIndex = splitPosition + 1;
	}

	// Set parent nodes.
	parents[leftIndex] = ~idx;
	parents[rightIndex] = idx;

	// Set the parent of the root node to -1.
	if (idx == 0)
	{
		parents[0] = 0xEFFFFFFF;
	}
}

// Note: dataIndices is of length numPrimitives.
template <typename T> __global__ void build_lbvh_treeD(
	u32 numPrimitives,
	T* sortedKeys, 
	i32 *parents
)
{
	const i32 idx = threadIdx.x + blockIdx.x * blockDim.x;

	// Check for valid threads.
	if (idx >= (numPrimitives - 1))
	{
		return;
	}

	build_lbvh_tree<T>(numPrimitives, sortedKeys, parents, idx);
}

struct BBCache {
	ei::Box bb;
	float cost;
	i32 primCount;
};
static_assert(sizeof(BBCache) == 8*sizeof(float), "Alignment of BBCache will be broken.");

template < Device dev >
CUDA_FUNCTION void calculate_bounding_boxes(
	const ObjectDescriptor<dev>& obj,
	ei::Vec4 * __restrict__ boundingBoxes, //TODO remove __restricts?
	i32 *sortedIndices,
	i32 * __restrict__ parents,
	i32 *collapseOffsets,
	const ei::Vec4& traversalCosts,
	i32* counters,
	const i32 idx,
	const i32 firstThreadInBlock = 0,
	const i32 lastThreadInBlock = 0,
	BBCache* sharedBb = nullptr
) {
	// If node is to be collapsed, offsets, dataIndices need to be updated.
	bool checkCollapse = true;
	i32 primId = sortedIndices[idx];

	// Calculate leaves bounding box and set primitives count and intersection test cost.
	ei::Box currentBb;
	float cost(traversalCosts.x), costAsLeaf;
	// Some auxilary variables for calculating primitiveCount.
	ei::IVec4 counts = { 0,0,0,1 }; // x: tri; y: quad; z: sphere; w: total.
	ei::IVec4 otherCounts; // x: tri; y: quad; z: sphere; w: total.
	// primitiveCount stores the numbers of each primitives in form:
	// 2 unused bits + 10 bits triangle + 10 bits quads + 10 bits spheres.
	i32 primitiveCount;
	if (primId >= i32(obj.polygon.numTriangles + obj.polygon.numQuads)) {
		primitiveCount = 1;
		cost += traversalCosts.w;
		counts.z = 1;
	} else if (primId >= i32(obj.polygon.numTriangles)) {
		primitiveCount = 0x00000400;
		cost += traversalCosts.z;
		counts.y = 1;
	} else {
		primitiveCount = 0x00100000;
		cost += traversalCosts.y;
		counts.x = 1;
	}
	currentBb = get_bounding_box(obj, primId);

	// Store cost and primitiveCount.
	cost *= ei::surface(currentBb);

	// Update node bounding boxes of current node.
#ifdef __CUDA_ARCH__
	sharedBb[threadIdx.x] = { currentBb, cost, primitiveCount };
#endif
	const i32 numInternalNodes = obj.numPrimitives - 1;
	i32 leafIndex = idx + numInternalNodes;
	i32 boxId = leafIndex << 1;
	boundingBoxes[boxId] = { currentBb.min, int_bits_as_float(primitiveCount) };
	boundingBoxes[boxId + 1] = { currentBb.max, cost };

	syncthreads(); // TODO check this.

	// Initialize.
	i32 current = parents[leafIndex];
	bool lastNodeIsLeftChild = false;
	if (current < 0) {
		current = ~current;
		lastNodeIsLeftChild = true;
	}
	i32 lastNode = idx; // Does not need to be initialized, since leaves will not be collapsed
				  // due to positive values of primitiveCount

			// In the counters array, we have stored the id of the thread that processed the other
		// children of this node.
#ifdef __CUDA_ARCH__
	u32 childThreadId = atomicExch(&counters[current], leafIndex);
#else
	i32 childThreadId = counters[current];
	counters[current] = leafIndex;
#endif // __CUDA_ARCH__

	// The first thread to reach a node will just die.
	if (childThreadId == 0xFFFFFFFF)
	{
		return;
	}

	while (true)
	{
		// Fetch bounding boxes and counts information.
		BBCache childInfo;
		i32 anotherChildId = (lastNodeIsLeftChild) ? lastNode + 1 : lastNode - 1;
		if (childThreadId >= numInternalNodes) {
			childThreadId -= numInternalNodes;
			anotherChildId += numInternalNodes;
		}
#ifdef __CUDA_ARCH__
		if (childThreadId >= firstThreadInBlock && childThreadId <= lastThreadInBlock) {
			// If both child nodes were processed by the same block, we can reuse the values
			// cached in shared memory.
			i32 childThreadIdInBlock = childThreadId - firstThreadInBlock;
			childInfo = sharedBb[childThreadIdInBlock];
		}
		else {
			// The children were processed in different blocks, so we have to find out if the one
			// that was not processed by this thread was the left or right one.
			boxId = anotherChildId << 1;
			ei::Vec4 childBbMin = boundingBoxes[boxId];
			ei::Vec4 childBbMax = boundingBoxes[boxId + 1];
			childInfo = BBCache{
				ei::Box{ei::Vec3{childBbMin}, ei::Vec3{childBbMax}},
				childBbMax.w, float_bits_as_int(childBbMin.w)
			};
		}
#else
		boxId = anotherChildId << 1;
		ei::Vec4 childBbMin = boundingBoxes[boxId];
		ei::Vec4 childBbMax = boundingBoxes[boxId + 1];
		childInfo = BBCache{
			ei::Box{ei::Vec3{childBbMin}, ei::Vec3{childBbMax}},
			childBbMax.w, float_bits_as_int(childBbMin.w)
		};
#endif // __CUDA_ARCH__
		syncthreads(); // @todo check.

		currentBb = ei::Box{ currentBb, childInfo.bb };
		if (checkCollapse) {
			// Calculate primitves counts.
			// Set offsets.
			if (primitiveCount < 0) { // Count < 0 means the node should be collapsed.
				primitiveCount &= 0x7FFFFFFF;
				// offset is the number of internal nodes below the child lastNode.
				i32 offset = counts.w - 2;
				if (lastNodeIsLeftChild)
					collapseOffsets[lastNode] = offset;
				else
					// Since lastNode as right child should be collapsed, lastNode + 1
					// must be one child of lastNode if it has more than 2 primitves.
					collapseOffsets[lastNode + 1] = offset;
			}

			extract_prim_counts(childInfo.primCount, otherCounts);

			if (childInfo.primCount < 0) {
				//childInfo.primCount &= 0x7FFFFFFF; // does not needed due to & 0x3FFFFFFF.
				// offset is the number of internal nodes below the other child.
				i32 offset = otherCounts.w - 2;
				if (lastNodeIsLeftChild)
					collapseOffsets[anotherChildId + 1] = offset;
				else
					// Since theOtherChild as right child should be collapsed, lastNode + 1
					// must be one child of theOtherChild if it has more than 2 primitves.
					collapseOffsets[anotherChildId] = offset;
			}

			// Update primtivesCount.
			counts.x += otherCounts.x;
			counts.y += otherCounts.y;
			counts.z += otherCounts.z;

			if (counts.x > 1023 || counts.y > 1023 || counts.z > 1023) {
				checkCollapse = false;
				// Setting cacheMin.w is here to make sure:
				// even if the current node has checkCollapse = false but be killed in 
				// the next round, however, primitiveCount will be read to disable checkCollapse.
				primitiveCount = 0x00000FFF;
			} else {
				// & 0x3FFFFFFF is used to avoid intervention of the leftChildMarkBit.
				primitiveCount += (childInfo.primCount & 0x3FFFFFFF);
				// Calculate costs.
				float area = ei::surface(currentBb);
				cost = traversalCosts.x * area + childInfo.cost + cost;
				counts.w += otherCounts.w;// Determine offset.
				costAsLeaf = area * (counts.x * traversalCosts.y + counts.y * traversalCosts.z + counts.z * traversalCosts.w);
				if (costAsLeaf < cost) {
					// Collapse.
					primitiveCount |= 0x80000000;
					// Update cost.
					cost = costAsLeaf;
				}
			}
		}

		// Update last processed node
		lastNode = current;

		// Update current node pointer
		current = parents[current];
		// If current == 0, both left/right children are taken as left children
		// for setting offset if needed.
		if (current < 0) {
			current = ~current;
			lastNodeIsLeftChild = true;
		}
		else {
			lastNodeIsLeftChild = false;
		}

		// Update node bounding box of the last node.
		// Put this operation here because we need to 
		// mark the 2. highest bit of cacheMin.w as 1
		// is the lastNode is left child, else mark as 0.
		// This is for simplifying mark_nodesD.
		if (checkCollapse) {
			if (lastNodeIsLeftChild)
				primitiveCount |= 0x40000000;
			else
				primitiveCount &= 0xBFFFFFFF;
		}

#ifdef __CUDA_ARCH__
		sharedBb[threadIdx.x] = BBCache{ currentBb, cost, primitiveCount };
#endif // __CUDA_ARCH__

		boxId = lastNode << 1;
		boundingBoxes[boxId] = { currentBb.min, int_bits_as_float(primitiveCount) };
		boundingBoxes[boxId + 1] = { currentBb.max, cost };

		syncthreads(); //@todo check.

		if (lastNode == 0) {
			// Print the bounding box of the base node.
			printf("root bounding box:\n%f %f %f\n%f %f %f\n",
				currentBb.min.x, currentBb.min.y, currentBb.min.z,
				currentBb.max.x, currentBb.max.y, currentBb.max.z);
			return;
		}

		// In the counters array, we have stored the id of the thread that processed the other
// children of this node.
#ifdef __CUDA_ARCH__
		childThreadId = atomicExch(&counters[current], idx);
#else
		childThreadId = counters[current];
		counters[current] = idx;
#endif // __CUDA_ARCH__

		// The first thread to reach a node will just die.
		if (childThreadId == 0xFFFFFFFF)
		{
			return;
		}
	}
}

__global__ void calculate_bounding_boxesD(
	const ObjectDescriptor<Device::CUDA>& obj,
	ei::Vec4* __restrict__ boundingBoxes, //TODO remove __restricts?
	i32* sortedIndices,
	i32* __restrict__ parents,
	i32* collapseOffsets,
	const ei::Vec4& traversalCosts,
	i32* counters)
{
	const i32 idx = threadIdx.x + blockIdx.x * blockDim.x;
	const i32 firstThreadInBlock = blockIdx.x * blockDim.x;
	const i32 lastThreadInBlock = firstThreadInBlock + blockDim.x - 1;

	// Initialize cache of bounding boxes in shared memory.
	extern __shared__ BBCache sharedBb[]; 

	// Check for valid threads.
	if (idx >= obj.numPrimitives)
		return;

	calculate_bounding_boxes(obj, boundingBoxes, sortedIndices, parents,
		collapseOffsets, traversalCosts, counters, idx,
		firstThreadInBlock, lastThreadInBlock, sharedBb);
}

template < Device dev >
CUDA_FUNCTION void calculate_bounding_boxes_ins(
	const SceneDescriptor<dev>& scene,
	ei::Vec4 * __restrict__ boundingBoxes, //TODO remove __restricts?
	i32 *sortedIndices,
	i32 * __restrict__ parents,
	i32 *collapseOffsets,
	const float ci, 
	const float ct,
	i32* counters,
	const i32 idx,
	const i32 firstThreadInBlock = 0,
	const i32 lastThreadInBlock = 0,
	BBCache* sharedBb = nullptr) {
	// If node is to be collapsed, offsets, dataIndices need to be updated.
	bool checkCollapse = true;

	// Calculate leaves bounding box and set primitives count and intersection test cost.
	float costAsLeaf;
	const i32 objId = scene.objectIndices[idx];
	ei::Box currentBb = ei::transform(scene.aabbs[objId], scene.transformations[idx]);
	// Some auxilary variables for calculating primitiveCount.
	i32 instanceCount = 1;
	float cost = ci + ct;

	// Store cost and primitiveCount.
	cost *= ei::surface(currentBb);

	// Update node bounding boxes of current node.
#ifdef __CUDA_ARCH__
	sharedBb[threadIdx.x] = { currentBb, cost, instanceCount };
#endif
	const i32 numInternalNodes = scene.numInstances - 1;
	i32 leafIndex = idx + numInternalNodes;
	i32 boxId = leafIndex << 1;
	boundingBoxes[boxId] = { currentBb.min, int_bits_as_float(instanceCount) };
	boundingBoxes[boxId + 1] = { currentBb.max, cost };

	syncthreads(); // TODO check this.

	// Initialize.
	i32 current = parents[leafIndex];
	bool lastNodeIsLeftChild = false;
	if (current < 0) {
		current = ~current;
		lastNodeIsLeftChild = true;
	}
	i32 lastNode = idx; // Does not need to be initialized, since leaves will not be collapsed
				  // due to positive values of primitiveCount

			// In the counters array, we have stored the id of the thread that processed the other
		// children of this node.
#ifdef __CUDA_ARCH__
	u32 childThreadId = atomicExch(&counters[current], leafIndex);
#else
	i32 childThreadId = counters[current]; 
	counters[current] = leafIndex;
#endif // __CUDA_ARCH__

	// The first thread to reach a node will just die.
	if (childThreadId == 0xFFFFFFFF)
	{
		return;
	}

	while (true)
	{
		// Fetch bounding boxes and counts information.
		BBCache childInfo;
		i32 anotherChildId = (lastNodeIsLeftChild) ? lastNode + 1 : lastNode - 1;
		if (childThreadId >= numInternalNodes) {
			childThreadId -= numInternalNodes;
			anotherChildId += numInternalNodes;
		}
#ifdef __CUDA_ARCH__
		if (childThreadId >= firstThreadInBlock && childThreadId <= lastThreadInBlock) {
			// If both child nodes were processed by the same block, we can reuse the values
			// cached in shared memory.
			i32 childThreadIdInBlock = childThreadId - firstThreadInBlock;
			childInfo = sharedBb[childThreadIdInBlock];
		}
		else {
			// The children were processed in different blocks, so we have to find out if the one
			// that was not processed by this thread was the left or right one.
			boxId = anotherChildId << 1;
			ei::Vec4 childBbMin = boundingBoxes[boxId];
			ei::Vec4 childBbMax = boundingBoxes[boxId + 1];
			childInfo = BBCache{
				ei::Box{ei::Vec3{childBbMin}, ei::Vec3{childBbMax}},
				childBbMax.w, float_bits_as_int(childBbMin.w)
			};
		}
#else
		boxId = anotherChildId << 1;
		ei::Vec4 childBbMin = boundingBoxes[boxId];
		ei::Vec4 childBbMax = boundingBoxes[boxId + 1];
		childInfo = BBCache{
			ei::Box{ei::Vec3{childBbMin}, ei::Vec3{childBbMax}},
			childBbMax.w, float_bits_as_int(childBbMin.w)
		};
#endif // __CUDA_ARCH__
		syncthreads(); // @todo check.
		
		currentBb = ei::Box{ currentBb, childInfo.bb };
		if (checkCollapse) {
			// Calculate primitves counts.
			// Set offsets.
			if (instanceCount < 0) { // Count < 0 means the node should be collapsed.
				instanceCount &= 0x3FFFFFFF;
				// offset is the number of internal nodes below the child lastNode.
				i32 offset = instanceCount - 2;
				if (lastNodeIsLeftChild)
					collapseOffsets[lastNode] = offset;
				else
					// Since lastNode as right child should be collapsed, lastNode + 1
					// must be one child of lastNode if it has more than 2 primitves.
					collapseOffsets[lastNode + 1] = offset;
			}

			if (childInfo.primCount < 0) {
				// & 0x3FFFFFFF is used to avoid intervention of the leftChildMarkBit.
				childInfo.primCount &= 0x3FFFFFFF;
				// offset is the number of internal nodes below the other child.
				i32 offset = childInfo.primCount - 2;
				if (lastNodeIsLeftChild)
					collapseOffsets[anotherChildId + 1] = offset;
				else
					// Since theOtherChild as right child should be collapsed, lastNode + 1
					// must be one child of theOtherChild if it has more than 2 primitves.
					collapseOffsets[anotherChildId] = offset;
			}

			// Update primtivesCount.
			instanceCount += childInfo.primCount;
			instanceCount &= 0x3FFFFFFF;

			if (instanceCount > 1023) {
				checkCollapse = false;
				// Setting cacheMin.w is here to make sure:
				// even if the current node has checkCollapse = false but be killed in 
				// the next round, however, primitiveCount will be read to disable checkCollapse.
				instanceCount = 0x00000FFF;
			}
			else {
				// Calculate costs.
				float area = ei::surface(currentBb);
				cost = ci * area + childInfo.cost + cost;
				costAsLeaf = area * instanceCount * ct;
				if (costAsLeaf < cost) {
					// Collapse.
					instanceCount |= 0x80000000;
					// Update cost.
					cost = costAsLeaf;
				}
			}
		}

		// Update last processed node
		lastNode = current;

		// Update current node pointer
		current = parents[current];
		// If current == 0, both left/right children are taken as left children
		// for setting offset if needed.
		if (current < 0) {
			current = ~current;
			lastNodeIsLeftChild = true;
		}
		else {
			lastNodeIsLeftChild = false;
		}

		// Update node bounding box of the last node.
		// Put this operation here because we need to 
		// mark the 2. highest bit of cacheMin.w as 1
		// is the lastNode is left child, else mark as 0.
		// This is for simplifying mark_nodesD.
		if (checkCollapse) {
			if (lastNodeIsLeftChild)
				instanceCount |= 0x40000000;
			else
				instanceCount &= 0xBFFFFFFF;
		}

#ifdef __CUDA_ARCH__
		sharedBb[threadIdx.x] = BBCache{ currentBb, cost, instanceCount };
#endif // __CUDA_ARCH__

		boxId = lastNode << 1;
		boundingBoxes[boxId] = { currentBb.min, int_bits_as_float(instanceCount) };
		boundingBoxes[boxId + 1] = { currentBb.max, cost };

		syncthreads(); //@todo check.

		if (lastNode == 0) {
			// Print the bounding box of the base node.
			printf("root bounding box:\n%f %f %f\n%f %f %f\n",
				currentBb.min.x, currentBb.min.y, currentBb.min.z,
				currentBb.max.x, currentBb.max.y, currentBb.max.z);
			return;
		}

		// In the counters array, we have stored the id of the thread that processed the other
// children of this node.
#ifdef __CUDA_ARCH__
		childThreadId = atomicExch(&counters[current], idx);
#else
		childThreadId = counters[current];
		counters[current] = idx;
#endif // __CUDA_ARCH__
		

		// The first thread to reach a node will just die.
		if (childThreadId == 0xFFFFFFFF)
		{
			return;
		}
	}
}

__global__ void calculate_bounding_boxes_insD(
	const SceneDescriptor<Device::CUDA>& scene,
	ei::Vec4 * __restrict__ boundingBoxes, //TODO remove __restricts?
	i32 *sortedIndices,
	i32 * __restrict__ parents,
	i32 *collapseOffsets,
	float ci, float ct,
	i32* counters)
{
	const i32 idx = threadIdx.x + blockIdx.x * blockDim.x;
	const i32 firstThreadInBlock = blockIdx.x * blockDim.x;
	const i32 lastThreadInBlock = firstThreadInBlock + blockDim.x - 1;

	// Initialize cache of bounding boxes in shared memory.
	extern __shared__ BBCache sharedBb[];

	// Check for valid threads.
	if (idx >= scene.numInstances)
		return;

	calculate_bounding_boxes_ins(scene, boundingBoxes,
		sortedIndices, parents, collapseOffsets, ci, ct, counters, idx, 
		firstThreadInBlock, lastThreadInBlock, sharedBb);
}

CUDA_FUNCTION void mark_nodes(
	u32 numInternalNodes,
	const ei::Vec4* __restrict__ boundingBoxes,
	i32* __restrict__ removedMarks,
	i32* __restrict__ reduceOffsets,
	i32* __restrict__ leafMarks,
	const i32 idx
) {
	// This is better than using a stack, since less operations are needed
	// and due to parallerism, performance is not affected.
	const i32 boxId = idx << 1;
	i32 primitiveCount = float_bits_as_int(boundingBoxes[boxId].w);
	if (primitiveCount >= 0) {
		leafMarks[idx] = 0;
		return;
	}
	leafMarks[idx] = 1;

	ei::IVec4 count;
	// Extract counts for three kinds of primitvies.
	extract_prim_counts(primitiveCount, count);
	//if (count.w == 2) // Not needed due to start <= end check.
	//	return; 
	i32 start, end;
	if ((primitiveCount & 0x40000000) == 0) {
		// Current node is a right child.
		start = idx + 1;
		end = idx + count.w - 2;
	}
	else {
		// Current node is a left child.
		end = idx - 1;
		start = idx - count.w + 2;
	}
	while (start <= end) {
		removedMarks[start] = 0xFFFFFFFF;
		reduceOffsets[start] = 0;
		++start;
	}
}

__global__ void mark_nodesD(
	u32 numInternalNodes,
	const ei::Vec4* __restrict__ boundingBoxes,
	i32* __restrict__ removedMarks,
	i32* __restrict__ reduceOffsets,
	i32* __restrict__ leafMarks
) {
	const i32 idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx >= numInternalNodes)
		return;

	mark_nodes(numInternalNodes, boundingBoxes, removedMarks, reduceOffsets, leafMarks, idx);
}


CUDA_FUNCTION i32 insert_id(i32 id, const i32* preLeaves, i32 removedLeaves) {
	i32 numPreLeaves = preLeaves[id];
	return (id << 2) - numPreLeaves * 3 - removedLeaves;
}

CUDA_FUNCTION i32 next_id(ei::IVec4& insertPos, ei::IVec2& endPos, i32& primType) {
	if (insertPos.x < endPos.x) {
		primType = 0;
		return insertPos.x;
	}
	else if (insertPos.y < endPos.y) {
		primType = 1;
		return insertPos.y;
	}
	else {
		primType = 2;
		return insertPos.z;
	}
}

CUDA_FUNCTION void copy_to_collapsed_bvh(
	const i32 numNodes,
	const i32 numInternalNodes,
	const i32 simpleLeafOffset,
	ei::Vec4* collapsedBVH,
	const ei::Vec4 * __restrict__ boundingBoxes,
	const i32* __restrict__ parents,
	const i32* __restrict__ removedMarks,
	i32* __restrict__ sortIndices,
	const i32* __restrict__ preInternalLeaves,
	const i32* __restrict__ reduceOffsets,
	const i32 offsetQuads, const i32 offsetSpheres,
	const i32 idx
) {
	if (idx >= numInternalNodes) {
		i32 parent = parents[idx];
		i32 offset = 1;
		if (parent < 0) {// Is left child.
			parent = ~parent;
			offset = 0;
		}

		const i32 boxId = parent << 1;
		i32 primitiveCount = float_bits_as_int(boundingBoxes[boxId].w);
		if (primitiveCount >= 0 && removedMarks[parent] != 0xFFFFFFFF) {

			// Copy bounding boxes to parent.
			// Set to parent id in collapsed bvh.
			const i32 removedLeaves = reduceOffsets[parent];
			i32 nodeId = idx << 1;
			ei::Vec4 lo = boundingBoxes[nodeId];
			ei::Vec4 hi = boundingBoxes[nodeId + 1];
			const i32 insertId = insert_id(parent, preInternalLeaves, removedLeaves);
			collapsedBVH[insertId + offset] = ei::Vec4(lo.x, hi.x, lo.y, hi.y);
			*(((ei::Vec2*)&collapsedBVH[insertId + 2]) + offset) = ei::Vec2(lo.z, hi.z);

			// Set child pointers to parent.
			// set to current node id in collapsed bvh.
			*(((i32*)&collapsedBVH[insertId + 3]) + offset) =
				~(simpleLeafOffset + idx);
		}

	} else
	if (removedMarks[idx] != 0xFFFFFFFF) {
		i32 parent = parents[idx];
		i32 offset = 1;
		if (parent < 0) {// Is left child.
			parent = ~parent;
			offset = 0;
		}

		// Copy bounding boxes to parent.
		// Set to parent id in collapsed bvh.
		i32 removedInternalLeaves = reduceOffsets[parent];
		i32 nodeId = idx << 1;
		const ei::Vec4 lo = boundingBoxes[nodeId];
		const ei::Vec4 hi = boundingBoxes[nodeId + 1];
		const i32 insertId = insert_id(parent, preInternalLeaves, removedInternalLeaves);
		collapsedBVH[insertId + offset] = ei::Vec4(lo.x, hi.x, lo.y, hi.y);
		*(((ei::Vec2*)&collapsedBVH[insertId + 2]) + offset) = ei::Vec2(lo.z, hi.z);

		// Set child pointers to parent.
		// set to current node id in collapsed bvh.
		removedInternalLeaves = reduceOffsets[idx];
		int pointerId = insert_id(idx, preInternalLeaves, removedInternalLeaves);
		// Leaf with negative sign to the pointer.
		*(((i32*)&collapsedBVH[insertId + 3]) + offset) = (lo.w < 0) ? ~pointerId: pointerId;
		
		// Copy data indices.
		if (lo.w < 0) {
			i32 primitiveCount = float_bits_as_int(lo.w);
			ei::IVec4 counts;
			extract_prim_counts(primitiveCount, counts);
			i32 startId;
			if (offset) { // The current node is right child.
				startId = idx;
			}
			else {
				startId = idx - counts.w + 1;
			}

			// Now counts is used to register insert position.
			ei::IVec2 endPos;
			endPos.x = startId + counts.x;
			endPos.y = endPos.x + counts.y;
			i32 primId;
			i32 nextId;
			i32 primInsertId;
			bool readId = true;
			counts.x = startId;
			counts.y = endPos.x;
			counts.z = endPos.y;
			collapsedBVH[pointerId] = ei::Vec4(
				lo.w, int_bits_as_float(counts.x),
				int_bits_as_float(counts.y), int_bits_as_float(counts.z));
			i32 primType; // 0: tri; 1: quad; 2: sph.
			i32 tmpId;
			// Use a loop to set data indices.
			// TODO: try simple read/write version to reduce branches.
			while (counts.w != 0) {
				if (readId) {
					nextId = next_id(counts, endPos, primType);
					primId = sortIndices[nextId];

					if (primId < offsetQuads) {
						// Read a triangle.
						if (primType == 0) {
							goto stay_at_place;
						}
						primType = 0;
					}
					else if (primId < offsetSpheres) {
						// Read a quad.
						if (primType == 1) {
							goto stay_at_place;
						}
						primType = 1;
					}
					else { // Read a sphere.
						if (primType == 2) {
							goto stay_at_place;
						}
						primType = 2;
					}
					readId = false;
				}

				//Now primType matches current primId.
				primInsertId = counts[primType]++;
				--counts.w;

				if ((primInsertId == nextId)) {
					readId = true;
				}
				else {
					tmpId = sortIndices[primInsertId];
					if (tmpId < offsetQuads) {
						if (primType == 0) {
							continue;
						}
						primType = 0;
					}
					else if (tmpId < offsetSpheres) {
						if (primType == 1) {
							continue;
						}
						primType = 1;
					}
					else if (primType == 2) {
						continue;
					}
					else {
						primType = 2;
					}

				}
				sortIndices[primInsertId] = primId;
				primId = tmpId;
				continue;

			stay_at_place:
				--counts.w; // Reduce the numbe of to be inserted primitives.
				counts[primType]++;
			}
		}
	}
}

__global__ void copy_to_collapsed_bvhD(
	const i32 numNodes,
	const i32 numInternalNodes,
	const i32 simpleLeafOffset,
	ei::Vec4* collapsedBVH,
	const ei::Vec4 * __restrict__ boundingBoxes,
	const i32* __restrict__ parents,
	const i32* __restrict__ removedMarks,
	i32* __restrict__ sortIndices,
	const i32* __restrict__ preInternalLeaves,
	const i32* __restrict__ reduceOffsets,
	const i32 offsetQuads, const i32 offsetSpheres
) {
	i32 idx = threadIdx.x + blockIdx.x * blockDim.x + 1;

	if (idx >= numNodes)
		return;

	copy_to_collapsed_bvh(numNodes, numInternalNodes, simpleLeafOffset, collapsedBVH,
		boundingBoxes, parents, removedMarks, sortIndices, preInternalLeaves, reduceOffsets,
		offsetQuads, offsetSpheres, idx);
}

CUDA_FUNCTION void copy_to_collapsed_bvh_ins(
	i32 numNodes,
	i32 numInternalNodes,
	i32 simpleLeafOffset,
	ei::Vec4* collapsedBVH,
	ei::Vec4 * __restrict__ boundingBoxes,
	i32* __restrict__ parents,
	i32* __restrict__ removedMarks,
	i32* __restrict__ sortIndices,
	i32* __restrict__ preInternalLeaves,
	i32* __restrict__ reduceOffsets,
	const i32 idx
) {
	if (idx >= numInternalNodes) {
		i32 parent = parents[idx];
		i32 offset = 1;
		if (parent < 0) {// Is left child.
			parent = ~parent;
			offset = 0;
		}

		const i32 boxId = parent << 1;
		i32 primitiveCount = float_bits_as_int(boundingBoxes[boxId].w);
		if (primitiveCount >= 0 && removedMarks[parent] != 0xFFFFFFFF) {

			// Copy bounding boxes to parent.
			// Set to parent id in collapsed bvh.
			const i32 removedLeaves = reduceOffsets[parent];
			i32 nodeId = idx << 1;
			ei::Vec4 lo = boundingBoxes[nodeId];
			ei::Vec4 hi = boundingBoxes[nodeId + 1];
			const i32 insertId = insert_id(parent, preInternalLeaves, removedLeaves);
			collapsedBVH[insertId + offset] = ei::Vec4(lo.x, hi.x, lo.y, hi.y);
			*(((ei::Vec2*)&collapsedBVH[insertId + 2]) + offset) = ei::Vec2(lo.z, hi.z);

			// Set child pointers to parent.
			// set to current node id in collapsed bvh.
			*(((i32*)&collapsedBVH[insertId + 3]) + offset) =
				~(simpleLeafOffset + idx);
		}

	}
	else
		if (removedMarks[idx] != 0xFFFFFFFF) {
			i32 parent = parents[idx];
			i32 offset = 1;
			if (parent < 0) {// Is left child.
				parent = ~parent;
				offset = 0;
			}

			// Copy bounding boxes to parent.
			// Set to parent id in collapsed bvh.
			i32 removedInternalLeaves = reduceOffsets[parent];
			i32 nodeId = idx << 1;
			const ei::Vec4 lo = boundingBoxes[nodeId];
			const ei::Vec4 hi = boundingBoxes[nodeId + 1];
			const i32 insertId = insert_id(parent, preInternalLeaves, removedInternalLeaves);
			collapsedBVH[insertId + offset] = ei::Vec4(lo.x, hi.x, lo.y, hi.y);
			*(((ei::Vec2*)&collapsedBVH[insertId + 2]) + offset) = ei::Vec2(lo.z, hi.z);

			// Set child pointers to parent.
			// set to current node id in collapsed bvh.
			removedInternalLeaves = reduceOffsets[idx];
			int pointerId = insert_id(idx, preInternalLeaves, removedInternalLeaves);
			// Leaf with negative sign to the pointer.
			*(((i32*)&collapsedBVH[insertId + 3]) + offset) = (lo.w < 0) ? ~pointerId : pointerId;

			// Set the pointer to instances.
			// Copy data indices is no longer needed.
			if (lo.w < 0) {
				i32 instanceCount = float_bits_as_int(lo.w) & 0x3FFFFFFF;
				i32 startId;
				if (offset) { // The current node is right child.
					startId = idx;
				}
				else {
					startId = idx - instanceCount + 1;
				}
				collapsedBVH[pointerId] = ei::Vec4(
					instanceCount, int_bits_as_float(startId), 0.f, 0.f);
			}
		}
}

__global__ void copy_to_collapsed_bvh_insD(
	i32 numNodes,
	i32 numInternalNodes,
	i32 simpleLeafOffset,
	ei::Vec4* collapsedBVH,
	ei::Vec4 * __restrict__ boundingBoxes,
	i32* __restrict__ parents,
	i32* __restrict__ removedMarks,
	i32* __restrict__ sortIndices,
	i32* __restrict__ preInternalLeaves,
	i32* __restrict__ reduceOffsets
) {
	i32 idx = threadIdx.x + blockIdx.x * blockDim.x + 1;

	if (idx >= numNodes)
		return;


	copy_to_collapsed_bvh_ins(numNodes, numInternalNodes, simpleLeafOffset, collapsedBVH,
		boundingBoxes, parents, removedMarks, sortIndices, preInternalLeaves, reduceOffsets, idx);
}

//}} // namespace mufflon:: {
}}}


namespace mufflon { namespace scene { namespace accel_struct {

// For the objects.
template < Device dev >
void LBVHBuilder::build_lbvh(const ObjectDescriptor<dev>& obj,
							 const ei::Box& sceneBB,
							 const ei::Vec4& traverseCosts) {
	if(obj.numPrimitives == 1) {
		// TODO remove this. 
		m_primIds.resize(1);
		m_primIds.synchronize<dev>();
		m_bvhNodes.resize(1);
		m_bvhNodes.synchronize<dev>();
		return;
	}

	i32 numBlocks, numThreads;

	// Allocate memory for a part of the BVH.We do not know the final size yet and
	// cannot allocate the other parts in bvh.
	m_primIds.resize(obj.numPrimitives * sizeof(i32));
	i32* primIds = as<i32>(m_primIds.acquire<dev>());

	// Calculate Morton codes.
	auto codesMem = make_udevptr_array<dev, u64>(obj.numPrimitives);
	const i32 numInternalNodes = obj.numPrimitives - 1;
	const u32 numNodes = numInternalNodes + obj.numPrimitives;
	auto parentsMem = make_udevptr_array<dev, i32>(numNodes);
	i32 *parents = parentsMem.get(); // size numNodes.
	{
		u64* mortonCodes = codesMem.get();
		if(dev == Device::CUDA) {
			// Satisfy the compiler (this brach is never reached without having the correct type,
			// so the cast is effectless. However, the cleaner 'if constexpr' is not supported in cuda.
			auto dobj = reinterpret_cast<const ObjectDescriptor<Device::CUDA>&>(obj);
			get_maximum_occupancy(numBlocks, numThreads, obj.numPrimitives, calculate_morton_codes64D);
			calculate_morton_codes64D << < numBlocks, numThreads >> > (
				dobj, sceneBB, mortonCodes, primIds);
			cuda::check_error(hipGetLastError());

			// Sort based on Morton codes.
			CuLib::DeviceSort(obj.numPrimitives, &mortonCodes, &mortonCodes,
				&primIds, &primIds);
			cuda::check_error(hipGetLastError());
		} else {
			for (i32 idx = 0; idx < obj.numPrimitives; idx++)
			{
				mortonCodes[idx] = calculate_morton_code<ObjectDescriptor<dev>, u64>(obj, idx, sceneBB);
				primIds[idx] = idx;
			}

			// Sort based on Morton codes.
			thrust::sort_by_key(mortonCodes, mortonCodes + obj.numPrimitives, primIds);
		}

		// Create BVH.
		// Layout: first internal nodes, then leves.
		if (dev == Device::CUDA) {
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(build_lbvh_treeD<u64>), hipFuncCachePreferL1);
			get_maximum_occupancy(numBlocks, numThreads, numInternalNodes, build_lbvh_treeD<u64>);
			build_lbvh_treeD<u64> << <numBlocks, numThreads >> > (//TODO check <u64>
				obj.numPrimitives,
				mortonCodes,
				parents);
			cuda::check_error(hipGetLastError());
		}
		else {
			for (i32 idx = 0; idx < numInternalNodes; idx++)
			{
				build_lbvh_tree<u64>(obj.numPrimitives, mortonCodes, parents, idx);
			}
		}
	}

	// Calcualte bounding boxes and SAH.
	// Create atomic counters buffer.
	auto deviceCountersMem = make_udevptr_array<dev, i32>(numInternalNodes);
	i32* deviceCounters = deviceCountersMem.get();
	mem_set<dev>(deviceCounters, 0xFF, numInternalNodes * sizeof(i32));
	// Allocate bounding boxes.
	auto boundingBoxes = make_udevptr_array<dev, ei::Vec4>(numNodes * 2);
	// Allocate collapseOffsets.
	// The last position for collapseOffsets is to avoid access violations,
	// since if the last internal node needs to be collapsed, it will write 
	// to this positions with offset = 0, but this info will not be used further.
	auto collapseOffsetsMem = make_udevptr_array<dev, i32>(numInternalNodes);
	i32* collapseOffsets = collapseOffsetsMem.get();
	mem_set<dev>(collapseOffsets, 0, numInternalNodes * sizeof(i32));
	i32* leafMarks = (i32*)codesMem.get();

	if (dev == Device::CUDA) {
		// Calculate BVH bounding boxes.
		i32 bboxCacheSize = numThreads * sizeof(ei::Vec4) * 2;
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(calculate_bounding_boxesD), hipFuncCachePreferShared);
		BoundingBoxFunctor functor;
		get_maximum_occupancy_variable_smem(numBlocks, numThreads, obj.numPrimitives,
			calculate_bounding_boxesD, functor);
		calculate_bounding_boxesD << <numBlocks, numThreads, bboxCacheSize >> > (
			reinterpret_cast<const ObjectDescriptor<Device::CUDA>&>(obj), // See above
			boundingBoxes.get(),
			primIds,
			parents,
			collapseOffsets,
			traverseCosts,
			deviceCounters);

		// Mark all children of collapsed nodes as removed and themselves as leaves (=1).
		get_maximum_occupancy(numBlocks, numThreads, numInternalNodes, mark_nodesD);
		mark_nodesD << <numBlocks, numThreads, bboxCacheSize >> > (
			numInternalNodes,
			boundingBoxes.get(),
			deviceCounters,
			collapseOffsets,
			leafMarks);
	} else {
		// Calculate BVH bounding boxes.
		for (i32 idx = 0; idx < obj.numPrimitives; idx++)
		{
			calculate_bounding_boxes(obj,
				boundingBoxes.get(), primIds, parents, collapseOffsets,
				traverseCosts, deviceCounters, idx);
		}

		// Mark all children of collapsed nodes as removed and themselves as leaves (=1).
		for (i32 idx = 0; idx < numInternalNodes; idx++)
		{
			mark_nodes(numInternalNodes, boundingBoxes.get(), deviceCounters, collapseOffsets, leafMarks, idx);
		}
	}

	i32 numRemovedInternalNodes;
	if (dev == Device::CUDA) {
		// Scan to get values for offsets.
		// TODO: is i32 enough??? It might overflow in the sum. I do not know what collapseOffsets should contain at this point, so I connot judge.
		CuLib::DeviceInclusiveSum(numInternalNodes, collapseOffsets, collapseOffsets);
		copy(&numRemovedInternalNodes, collapseOffsets + numInternalNodes - 1, sizeof(i32));
		// Scan to get number of leaves arised from internal nodes before current node.
		CuLib::DeviceExclusiveSum(numInternalNodes + 1, leafMarks, leafMarks);
	}
	else {
		// Scan to get values for offsets.
		thrust::inclusive_scan(collapseOffsets, collapseOffsets + numInternalNodes,
			collapseOffsets);
		numRemovedInternalNodes = collapseOffsets[numInternalNodes - 1];
		// Scan to get number of leaves arised from internal nodes before current node.
		thrust::exclusive_scan(leafMarks, leafMarks + numInternalNodes + 1, leafMarks);
	}

	// Here uses a compact memory layout so that each leaf only uses 16 bytes.
	i32 numInternLeavesCollapsedBVH; //!= (numNodesCollapsedBVH + 1) >> 1; since there are simple leaves.
	// and it includes the removed ones.

	if (dev == Device::CUDA)
		hipMemcpy(&numInternLeavesCollapsedBVH, leafMarks + numInternalNodes, sizeof(i32),
			hipMemcpyDeviceToHost);
	else
		numInternLeavesCollapsedBVH = leafMarks[numInternalNodes];

	i32 numInternalNodesCollapsedBVH = numInternalNodes - numInternLeavesCollapsedBVH;
	numInternLeavesCollapsedBVH -= numRemovedInternalNodes;
	i32 numFloat4InCollapsedBVH = numInternLeavesCollapsedBVH + 4 * numInternalNodesCollapsedBVH;
	//printf("bvhSize %d %d %d\n", bvhSize, 
	//	numInternLeavesCollapsedBVH + numRemovedInternalNodes, numRemovedInternalNodes);
	m_bvhNodes.resize(numFloat4InCollapsedBVH * sizeof(ei::Vec4));
	ei::Vec4* collapsedBVH = as<ei::Vec4>(m_bvhNodes.acquire<dev>());
	if (dev == Device::CUDA) {
		get_maximum_occupancy(numBlocks, numThreads, numNodes, copy_to_collapsed_bvhD);
		copy_to_collapsed_bvhD << < numBlocks, numThreads >> > (
			numNodes,
			numInternalNodes,
			numFloat4InCollapsedBVH - numInternalNodes,
			collapsedBVH,
			boundingBoxes.get(),
			parents,
			deviceCounters,
			primIds,
			leafMarks,
			collapseOffsets,
			obj.polygon.numTriangles,
			obj.polygon.numTriangles + obj.polygon.numQuads);
	} else {
		for (i32 idx = 1; idx < i32(numNodes); idx++)
		{
			copy_to_collapsed_bvh(numNodes, numInternalNodes, numFloat4InCollapsedBVH - numInternalNodes,
				collapsedBVH, boundingBoxes.get(), parents, deviceCounters, primIds,
				leafMarks, collapseOffsets, obj.polygon.numTriangles,
				obj.polygon.numTriangles + obj.polygon.numQuads, idx);
		}
	}
}

// For the scene.
template < Device dev >
void LBVHBuilder::build_lbvh(const SceneDescriptor<dev>& scene,
							 ei::Vec2 traverseCosts) {
	if(scene.numInstances == 1) {
		// TODO remove this. 
		m_primIds.resize(1);
		m_primIds.synchronize<dev>();
		m_bvhNodes.resize(1);
		m_bvhNodes.synchronize<dev>();
		return;
	}

	i32 numBlocks, numThreads;

	// Allocate memory for a part of the BVH. We do not know the final size yet and
	// cannot allocate the other parts in bvh.
	m_primIds.resize(scene.numInstances * sizeof(i32));
	i32* primIds = as<i32>(m_primIds.acquire<dev>());

	// Calculate Morton codes.
	auto codesMem = make_udevptr_array<dev, u32>(scene.numInstances);
	const i32 numInternalNodes = scene.numInstances - 1;
	const u32 numNodes = numInternalNodes + scene.numInstances;
	auto parentsMem = make_udevptr_array<dev, i32>(numNodes);
	i32 *parents = parentsMem.get(); // size numNodes.

	{
		u32* mortonCodes = codesMem.get();
		if (dev == Device::CUDA) {
			get_maximum_occupancy(numBlocks, numThreads, scene.numInstances, calculate_morton_codes32D);
			calculate_morton_codes32D << < numBlocks, numThreads >> > (
				reinterpret_cast<const SceneDescriptor<Device::CUDA>&>(scene),
				scene.aabb, mortonCodes, primIds);

			// Sort based on Morton codes.
			CuLib::DeviceSort(scene.numInstances, &mortonCodes, &mortonCodes,
				&primIds, &primIds);
		} else {
			for (i32 idx = 0; idx < scene.numInstances; idx++)
			{
				mortonCodes[idx] = calculate_morton_code<SceneDescriptor<dev>, u32>(scene, idx, scene.aabb);
				primIds[idx] = idx;
			}

			// Sort based on Morton codes.
			thrust::sort_by_key(mortonCodes, mortonCodes + scene.numInstances, primIds);
		}

		// Create BVH.
		// Layout: first internal nodes, then leves.
		if (dev == Device::CUDA) {
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(build_lbvh_treeD<u64>), hipFuncCachePreferL1);
			get_maximum_occupancy(numBlocks, numThreads, numInternalNodes, build_lbvh_treeD<u32>);
			build_lbvh_treeD<u32> << <numBlocks, numThreads >> > (//TODO check <u64>
				scene.numInstances,
				mortonCodes,
				parents);
		} else {
			for (i32 idx = 0; idx < numInternalNodes; idx++)
			{
				build_lbvh_tree<u32>(scene.numInstances, mortonCodes, parents, idx);
			}
		}
	}


	// Calcualte bounding boxes and SAH.
	// Create atomic counters buffer.
	auto deviceCountersMem = make_udevptr_array<dev, i32>(numInternalNodes);
	i32* deviceCounters = deviceCountersMem.get();
	mem_set<dev>(deviceCounters, 0xFF, numInternalNodes * sizeof(i32));
	// Allocate bounding boxes.
	auto boundingBoxes = make_udevptr_array<dev, ei::Vec4>(numNodes * 2);
	// Allocate collapseOffsets.
	// The last position for collapseOffsets is to avoid access violations,
	// since if the last internal node needs to be collapsed, it will write 
	// to this positions with offset = 0, but this info will not be used further.
	auto collapseOffsetsMem = make_udevptr_array<dev, i32>(numInternalNodes);
	i32* collapseOffsets = collapseOffsetsMem.get();
	mem_set<dev>(collapseOffsets, 0, numInternalNodes * sizeof(i32));
	i32* leafMarks = (i32*)codesMem.get();

	if (dev == Device::CUDA) {
		// Calculate BVH bounding boxes.
		i32 bboxCacheSize = numThreads * sizeof(ei::Vec4) * 2;
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(calculate_bounding_boxes_insD), hipFuncCachePreferShared);
		BoundingBoxFunctor functor;
		get_maximum_occupancy_variable_smem(numBlocks, numThreads, scene.numInstances,
			calculate_bounding_boxes_insD, functor);
		calculate_bounding_boxes_insD << <numBlocks, numThreads, bboxCacheSize >> > (
			reinterpret_cast<const SceneDescriptor<Device::CUDA>&>(scene),
			boundingBoxes.get(),
			primIds,
			parents,
			collapseOffsets,
			traverseCosts.x, traverseCosts.y,
			deviceCounters);

		// Mark all children of collapsed nodes as removed and themselves as leaves (=1).
		get_maximum_occupancy(numBlocks, numThreads, numInternalNodes, mark_nodesD);
		mark_nodesD << <numBlocks, numThreads, bboxCacheSize >> > (
			numInternalNodes,
			boundingBoxes.get(),
			deviceCounters,
			collapseOffsets,
			leafMarks);
	} else {
		// Calculate BVH bounding boxes.
		for (i32 idx = 0; idx < scene.numInstances; idx++)
		{
			calculate_bounding_boxes_ins(scene,
				boundingBoxes.get(),
				primIds,
				parents,
				collapseOffsets,
				traverseCosts.x, traverseCosts.y,
				deviceCounters,
				idx);
		}

		// Mark all children of collapsed nodes as removed and themselves as leaves (=1).
		for (i32 idx = 0; idx < numInternalNodes; idx++)
		{
			mark_nodes(numInternalNodes, boundingBoxes.get(), deviceCounters, collapseOffsets, leafMarks, idx);
		}
	}

	i32 numRemovedInternalNodes;
	if (dev == Device::CUDA) {
		// Scan to get values for offsets.
		CuLib::DeviceInclusiveSum(numInternalNodes, collapseOffsets, collapseOffsets);
		copy(&numRemovedInternalNodes, collapseOffsets + numInternalNodes - 1, sizeof(i32));
		// Scan to get number of leaves arised from internal nodes before current node.
		CuLib::DeviceExclusiveSum(numInternalNodes + 1, leafMarks, leafMarks);
	}
	else {
		// Scan to get values for offsets.
		thrust::inclusive_scan(collapseOffsets, collapseOffsets + numInternalNodes,
			collapseOffsets);
		numRemovedInternalNodes = collapseOffsets[numInternalNodes - 1];
		// Scan to get number of leaves arised from internal nodes before current node.
		thrust::exclusive_scan(leafMarks, leafMarks + numInternalNodes + 1, leafMarks);
	}

	// Here uses a compact memory layout so that each leaf only uses 16 bytes.
	i32 numInternLeavesCollapsedBVH; //!= (numNodesCollapsedBVH + 1) >> 1; since there are simple leaves.
	// and it includes the removed ones.

	if (dev == Device::CUDA)
		hipMemcpy(&numInternLeavesCollapsedBVH, leafMarks + numInternalNodes, sizeof(i32),
			hipMemcpyDefault);
	else
		numInternLeavesCollapsedBVH = leafMarks[numInternalNodes];

	i32 numInternalNodesCollapsedBVH = numInternalNodes - numInternLeavesCollapsedBVH;
	numInternLeavesCollapsedBVH -= numRemovedInternalNodes;
	i32 numFloat4InCollapsedBVH = numInternLeavesCollapsedBVH + 4 * numInternalNodesCollapsedBVH;
	//printf("bvhSize %d %d %d\n", bvhSize, 
	//	numInternLeavesCollapsedBVH + numRemovedInternalNodes, numRemovedInternalNodes);
	m_bvhNodes.resize(numFloat4InCollapsedBVH * sizeof(ei::Vec4));
	ei::Vec4* collapsedBVH = as<ei::Vec4>(m_bvhNodes.acquire<dev>());
	// Copy values for collapsed BVH.	
	if (dev == Device::CUDA) {
		get_maximum_occupancy(numBlocks, numThreads, numNodes, copy_to_collapsed_bvh_insD);
		copy_to_collapsed_bvh_insD << < numBlocks, numThreads >> > (
			numNodes,
			numInternalNodes,
			numFloat4InCollapsedBVH - numInternalNodes,
			collapsedBVH,
			boundingBoxes.get(),
			parents,
			deviceCounters,
			primIds,
			leafMarks,
			collapseOffsets
			);
	}
	else {
		for (i32 idx = 1; idx < (i32)numNodes; idx++)
		{
			copy_to_collapsed_bvh_ins(numNodes, numInternalNodes, numFloat4InCollapsedBVH - numInternalNodes, 
				collapsedBVH, boundingBoxes.get(), parents, deviceCounters, primIds, 
				leafMarks, collapseOffsets, idx);
		}
	}
}


template < Device dev >
void LBVHBuilder::build(ObjectDescriptor<dev>& obj, const ei::Box& aabb) {
	ei::Vec4 traverseCosts = { 1.0f, 1.2f, 2.4f, 1.f };
	build_lbvh<dev>(obj, aabb, traverseCosts);
	m_primIds.mark_changed(dev);
	m_bvhNodes.mark_changed(dev);
}

template void LBVHBuilder::build<Device::CPU>(ObjectDescriptor<Device::CPU>&, const ei::Box&);
template void LBVHBuilder::build<Device::CUDA>(ObjectDescriptor<Device::CUDA>&, const ei::Box&);

template < Device dev >
void LBVHBuilder::build(
	const SceneDescriptor<dev>& scene
) {
	ei::Vec2 traverseCosts = { 1.f, 20.f };// TODO: find value for this.
	build_lbvh<dev>(scene, traverseCosts);	
	m_primIds.mark_changed(dev);
	m_bvhNodes.mark_changed(dev);
}

template void LBVHBuilder::build<Device::CPU>(const SceneDescriptor<Device::CPU>&);
template void LBVHBuilder::build<Device::CUDA>(const SceneDescriptor<Device::CUDA>&);

}}} // namespace mufflon
