#include "hip/hip_runtime.h"
#include "intersection.hpp"
#include "accel_structs_commons.hpp"
#include "lbvh.hpp"
#include "util/types.hpp"

#include <hip/hip_runtime_api.h>
#include <ei/3dtypes.hpp>
#include <ei/3dintersection.hpp>

namespace mufflon { namespace scene { namespace accel_struct {

namespace {

#define STACK_SIZE              96 //64          // Size of the traversal stack in local memory.
#define OBJ_STACK_SIZE              64 //64          // Size of the traversal stack in local memory.
enum : i32 {
	EntrypointSentinel = (i32)0xFFFFFFFF,   // Bottom-most stack entry, indicating the end of traversal.
	IGNORE_ID = (i32)0xFFFFFFFF,
	SECOND_QUAD_TRIANGLE_BIT = (i32)0x80000000,
	SECOND_QUAD_TRIANGLE_MASK = (i32)0x7FFFFFFF,
};

// Experimentally determined best mix of float/i32/video minmax instructions for Kepler.
__device__ __inline__ i32   min_min(i32 a, i32 b, i32 c) { i32 v; asm("vmin.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ i32   min_max(i32 a, i32 b, i32 c) { i32 v; asm("vmin.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ i32   max_min(i32 a, i32 b, i32 c) { i32 v; asm("vmax.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ i32   max_max(i32 a, i32 b, i32 c) { i32 v; asm("vmax.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ float fmin_fmin(float a, float b, float c) { return __int_as_float(min_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmin_fmax(float a, float b, float c) { return __int_as_float(min_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmin(float a, float b, float c) { return __int_as_float(max_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmax(float a, float b, float c) { return __int_as_float(max_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float spanBeginKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d) { return fmax_fmax(fminf(a0, a1), fminf(b0, b1), fmin_fmax(c0, c1, d)); }
__device__ __inline__ float spanEndKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d) { return fmin_fmin(fmaxf(a0, a1), fmaxf(b0, b1), fmax_fmin(c0, c1, d)); }


CUDA_FUNCTION bool intersect(const ei::Vec3& boxMin, const ei::Vec3& boxMax,
	const ei::Vec3 invDir, const ei::Vec3 ood, 
	const float tmin, const float tmax, float& cmin) {//, float& cmax) {
#ifdef __CUDA_ARCH__
	ei::Vec3 lo = boxMin * invDir - ood;
	ei::Vec3 hi = boxMax * invDir - ood;
	cmin = spanBeginKepler(lo.x, hi.x, lo.y, hi.y, lo.z, hi.z, tmin);
	const float cmax = spanEndKepler(lo.x, hi.x, lo.y, hi.y, lo.z, hi.z, tmax);
	return cmin <= cmax;
#else
	// TODO: use the epsilon one? FastRay one?
	float t0 = boxMin.x * invDir.x - ood.x;
	float t1 = boxMax.x * invDir.x - ood.x;
	cmin = ei::min(t0, t1);
	float cmax = ei::max(t0, t1);
	if (cmax < tmin || cmin > tmax) return false;
	t0 = boxMin.y * invDir.y - ood.y;
	t1 = boxMax.y * invDir.y - ood.y;
	float min2 = ei::min(t0, t1);
	float max2 = ei::max(t0, t1);
	cmin = ei::max(cmin, min2);
	cmax = ei::min(cmax, max2);
	if (cmax < tmin || cmin > tmax || cmin > cmax) return false;
	t0 = boxMin.z * invDir.z - ood.z;
	t1 = boxMax.z * invDir.z - ood.z;
	min2 = ei::min(t0, t1);
	max2 = ei::max(t0, t1);
	cmin = ei::max(cmin, min2);
	cmax = ei::min(cmax, max2);
	return (cmax >= tmin) && (cmin <= tmax) && (cmin <= cmax);
#endif // __CUDA_ARCH__
}

template < Device dev >
CUDA_FUNCTION bool intersects_primitve(
	const ObjectDescriptor<dev>& obj,
	const ei::Ray& ray,
	const i32 primId,
	const i32 startPrimId,
	int& hitPrimId,
	float& hitT,				// In out: max hit distance before, if hit then returns the new distance
	ei::Vec3& hitBarycentric
) {
	if(primId < (i32)obj.polygon.numTriangles) {
		// Triangle.
		if(startPrimId == primId) return false; // Masking to avoid self intersections

		const ei::Vec3* meshVertices = obj.polygon.vertices;
		const i32 indexOffset = primId * 3;
		const ei::IVec3 ids = { obj.polygon.vertexIndices[indexOffset],
								obj.polygon.vertexIndices[indexOffset + 1],
								obj.polygon.vertexIndices[indexOffset + 2] };
		const ei::Triangle tri = { meshVertices[ids[0]],
								   meshVertices[ids[1]],
								   meshVertices[ids[2]] };

		float t;
		ei::Vec3 barycentric;
		if(ei::intersects(ray, tri, t, barycentric) && t < hitT) {
			hitT = t;
			hitBarycentric = barycentric;
			hitPrimId = primId;
			return true;
		}
	} else if(primId < (i32)(obj.polygon.numTriangles + obj.polygon.numQuads)) {
		// Quad.
		const i32 indexOffset = (primId - obj.polygon.numTriangles) * 4 + obj.polygon.numTriangles * 3;
		const ei::Vec3* meshVertices = obj.polygon.vertices;
		float t;
		ei::Vec3 barycentric;

		// Check first triangle
		if(startPrimId != primId) { // Masking to avoid self intersections
			const ei::IVec3 ids = { obj.polygon.vertexIndices[indexOffset],
									obj.polygon.vertexIndices[indexOffset + 1],
									obj.polygon.vertexIndices[indexOffset + 2] };
			const ei::Triangle tri = { meshVertices[ids[0]],
									   meshVertices[ids[1]],
									   meshVertices[ids[2]] };
			if(ei::intersects(ray, tri, t, barycentric) && t < hitT) {
				hitT = t;
				hitBarycentric = barycentric;
				hitPrimId = hitPrimId;
				return true;
			}
		}

		// Check second triangle
		if(startPrimId != (primId | SECOND_QUAD_TRIANGLE_BIT)) { // Masking to avoid self intersections
			const ei::IVec3 ids = { obj.polygon.vertexIndices[indexOffset],
									obj.polygon.vertexIndices[indexOffset + 2],
									obj.polygon.vertexIndices[indexOffset + 3] };
			const ei::Triangle tri = { meshVertices[ids[0]],
									   meshVertices[ids[1]],
									   meshVertices[ids[2]] };
			if(ei::intersects(ray, tri, t, barycentric) && t < hitT) {
				hitT = t;
				hitBarycentric = barycentric;
				hitPrimId = hitPrimId | SECOND_QUAD_TRIANGLE_BIT;
				return true;
			}
		}
	} else {
		// Sphere.
		if(startPrimId == primId) return false; // Masking to avoid self intersections
		const ei::Sphere& sph = obj.spheres.spheres[primId];
		float t;
		// TODO: use some epsilon?
		if(ei::intersects(ray, sph, t) && t < hitT) {
			hitT = t;
			hitPrimId = primId;
			// Barycentrics unused
			return true;
		}
	}
	return false;
}

} // namespace ::

template < Device dev >
CUDA_FUNCTION bool any_intersection_obj_lbvh_imp(
	const LBVH& bvh,
	const ObjectDescriptor<dev>& obj,
	const ei::Ray& ray,
	const i32 startPrimId,
	const ei::Vec3& invDir, 
	const ei::Vec3& ood,
	const float tmin,
	const float tmax,
	i32* traversalStack
) {
	// Since all threads go to the following branch if numPrimitives == 1,
	// there is no problem with branching.
	if(obj.numPrimitives == 1) {
		float hitT = tmax;
		ei::Vec3 hitBarycentric;
		i32 hitPrimitiveId;
		if(intersects_primitve(obj, ray, 0, startPrimId, hitPrimitiveId, hitT, hitBarycentric)) {
			return true;
		}
		return false;
	}

	// Setup traversal.
	traversalStack[0] = EntrypointSentinel;	// Bottom-most entry.
	i32 nodeAddr = 0; // Start from the root.  
	i32* stackPtr = traversalStack; // Current position in traversal stack.
	i32 primCount = 0; // Internal nodes have no primitives

	// Traversal loop.
	while(nodeAddr != EntrypointSentinel) {
		if(nodeAddr < bvh.numInternalNodes) { // Internal node?
			// Fetch AABBs of the two child bvh.
			i32 nodeIdx = nodeAddr * 4;
			const ei::Vec4 Lmin_cL = bvh.bvh[nodeIdx];
			const ei::Vec4 Lmax_nL = bvh.bvh[nodeIdx + 1];
			const ei::Vec4 Rmin_cR = bvh.bvh[nodeIdx + 2];
			const ei::Vec4 Rmax_nR = bvh.bvh[nodeIdx + 3];

			// Intersect the ray against the children bounds.
			float c0min, c1min;
			bool traverseChild0 = intersect(ei::Vec3{Lmin_cL}, ei::Vec3{Lmax_nL}, invDir, ood, tmin, tmax, c0min);
			bool traverseChild1 = intersect(ei::Vec3{Rmin_cR}, ei::Vec3{Rmax_nR}, invDir, ood, tmin, tmax, c1min);

			// Neither child was intersected => pop stack.
			if(!traverseChild0 && !traverseChild1) {
				nodeAddr = *stackPtr;
				--stackPtr;
				if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
					primCount = *stackPtr;
					--stackPtr;
				}
			}
			// Otherwise => fetch child pointers.
			else {
				nodeAddr = traverseChild0 ? float_bits_as_int(Lmin_cL.w) : float_bits_as_int(Rmin_cR.w);
				primCount = traverseChild0 ? float_bits_as_int(Lmax_nL.w) : float_bits_as_int(Rmax_nR.w);

				// Both children were intersected => push the farther one.
				if (traverseChild0 && traverseChild1) {
					i32 pushAddr = float_bits_as_int(Rmin_cR.w); // nodeAddr is Lmin_cL.w, this is the other one
					i32 pushCount = float_bits_as_int(Rmax_nR.w);
					if (c1min < c0min) {
						i32 tmp = nodeAddr;
						nodeAddr = pushAddr;
						pushAddr = tmp;
						tmp = primCount;
						primCount = pushCount;
						pushCount = tmp;
					}
					if(pushAddr >= bvh.numInternalNodes) { // Leaf? Then push the count too
						++stackPtr;
						*stackPtr = pushCount;
					}
					++stackPtr;
					*stackPtr = pushAddr;
				}
			}
		}

		if(nodeAddr >= bvh.numInternalNodes && nodeAddr != EntrypointSentinel) { // Leaf?
			const i32 primId = bvh.primIds[ nodeAddr - bvh.numInternalNodes ];

			// TODO: no loop here! better use only one 'primitive' and wait for the next while iteration
			for(i32 i = 0; i < primCount; i++) {
				float hitT = tmax;
				ei::Vec3 hitBarycentric;
				i32 hitPrimitiveId;
				if(intersects_primitve(obj, ray, primId, startPrimId, hitPrimitiveId, hitT, hitBarycentric))
					return true;
			}

			// Pop next node.
			nodeAddr = *stackPtr;
			--stackPtr;
			if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
				primCount = *stackPtr;
				--stackPtr;
			}
		}
	}
	return false;
}


template < Device dev >
CUDA_FUNCTION bool first_intersection_obj_lbvh_imp(
	const LBVH& bvh,
	const ObjectDescriptor<dev>& obj,
	const ei::Ray& ray,
	const i32 startPrimId,
	const ei::Vec3& invDir, 
	const ei::Vec3& ood,
	const float tmin,
	int& hitPrimId, float& hitT,
	ei::Vec3& hitBarycentric,
	i32* traversalStack
) {
	// Fast path - no BVH
	if(obj.numPrimitives == 1) {
		return intersects_primitve(obj, ray, 0, startPrimId,
			hitPrimId, hitT, hitBarycentric);
	}
	
	// Setup traversal.
	traversalStack[0] = EntrypointSentinel;	// Bottom-most entry.
	i32 nodeAddr = 0; // Start from the root.  
	i32* stackPtr = traversalStack; // Current position in traversal stack.
	i32 primCount = 0; // Internal nodes have no primitives

	bool hasHit = false;

	// Traversal loop.
	while(nodeAddr != EntrypointSentinel) {
		if(nodeAddr < bvh.numInternalNodes) { // Internal node?
			// Fetch AABBs of the two child bvh.
			i32 nodeIdx = nodeAddr * 4;
			const ei::Vec4 Lmin_cL = bvh.bvh[nodeIdx];
			const ei::Vec4 Lmax_nL = bvh.bvh[nodeIdx + 1];
			const ei::Vec4 Rmin_cR = bvh.bvh[nodeIdx + 2];
			const ei::Vec4 Rmax_nR = bvh.bvh[nodeIdx + 3];

			// Intersect the ray against the children bounds.
			float c0min, c1min;
			bool traverseChild0 = intersect(ei::Vec3{Lmin_cL}, ei::Vec3{Lmax_nL}, invDir, ood, tmin, hitT, c0min);
			bool traverseChild1 = intersect(ei::Vec3{Rmin_cR}, ei::Vec3{Rmax_nR}, invDir, ood, tmin, hitT, c1min);

			// Neither child was intersected => pop stack.
			if(!traverseChild0 && !traverseChild1) {
				nodeAddr = *stackPtr;
				--stackPtr;
				if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
					primCount = *stackPtr;
					--stackPtr;
				}
			}
			// Otherwise => fetch child pointers.
			else {
				nodeAddr = traverseChild0 ? float_bits_as_int(Lmin_cL.w) : float_bits_as_int(Rmin_cR.w);
				primCount = traverseChild0 ? float_bits_as_int(Lmax_nL.w) : float_bits_as_int(Rmax_nR.w);

				// Both children were intersected => push the farther one.
				if (traverseChild0 && traverseChild1) {
					i32 pushAddr = float_bits_as_int(Rmin_cR.w); // nodeAddr is Lmin_cL.w, this is the other one
					i32 pushCount = float_bits_as_int(Rmax_nR.w);
					if (c1min < c0min) {
						i32 tmp = nodeAddr;
						nodeAddr = pushAddr;
						pushAddr = tmp;
						tmp = primCount;
						primCount = pushCount;
						pushCount = tmp;
					}
					if(pushAddr >= bvh.numInternalNodes) { // Leaf? Then push the count too
						++stackPtr;
						*stackPtr = pushCount;
					}
					++stackPtr;
					*stackPtr = pushAddr;
				}
			}
		}

		if(nodeAddr >= bvh.numInternalNodes && nodeAddr != EntrypointSentinel) { // Leaf?
			const i32 primId = bvh.primIds[ nodeAddr - bvh.numInternalNodes ];

			// TODO: no loop here! better use only one 'primitive' and wait for the next while iteration
			for(i32 i = 0; i < primCount; i++) {
				if(intersects_primitve(obj, ray, primId, startPrimId, hitPrimId, hitT, hitBarycentric))
					hasHit = true;
			}

			// Pop next node.
			nodeAddr = *stackPtr;
			--stackPtr;
			if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
				primCount = *stackPtr;
				--stackPtr;
			}
		}
	}
	return hasHit;
}


template < Device dev > CUDA_FUNCTION
void first_intersection_scene_obj_lbvh(
	const SceneDescriptor<dev>& scene,
	const ei::Ray& ray,
	const RayIntersectionResult::HitID& startInsPrimId,
	const i32 instanceId,
	i32* traversalStack,
	float& hitT,
	i32& hitInstanceId,
	i32& hitPrimId,
	ei::Vec3& hitBarycentric
) {
	const ei::Mat3x3 invRotScale = ei::invert(ei::Mat3x3{scene.transformations[instanceId]});
	const ei::Vec3 invTranslation { -scene.transformations[instanceId][3],
									-scene.transformations[instanceId][7],
									-scene.transformations[instanceId][11] };
	ei::Ray transRay = { invRotScale * (ray.origin + invTranslation),
						 normalize(invRotScale * ray.direction) };
	const ei::Vec3 invDir = sdiv(1.0f, transRay.direction);
	const ei::Vec3 ood = transRay.origin * invDir;

	const i32 objId = scene.objectIndices[instanceId];
	const ei::Box& box = scene.aabbs[objId];
	const float tmin = 1e-6f * len(box.max - box.min);

	// Intersect the ray against the obj bounding box.
	float t;
	if(intersect(box.min, box.max, invDir, ood, tmin, hitT, t)) {
		// Intersect the ray against the obj primitive bvh.
		const ObjectDescriptor<dev>& obj = scene.objects[objId];
		const LBVH* lbvh = (LBVH*)obj.accelStruct.accelParameters;
		const i32 checkPrimId = (startInsPrimId.instanceId == instanceId) ? startInsPrimId.primId : IGNORE_ID;
		if(first_intersection_obj_lbvh_imp(
			*lbvh, obj, transRay, checkPrimId, invDir, ood, tmin,
			hitPrimId, hitT, hitBarycentric, traversalStack))
			hitInstanceId = instanceId;
	}
}

template < Device dev > __host__ __device__
RayIntersectionResult first_intersection_scene_lbvh_imp(
	const LBVH& bvh,
	const SceneDescriptor<dev>& scene,
	const ei::Ray ray,
	const RayIntersectionResult::HitID& startInsPrimId,
	const float tmax
) {
	const float tmin = 1e-7f * len(scene.aabb.max - scene.aabb.min);
	i32 hitPrimId = SECOND_QUAD_TRIANGLE_BIT;						// No primitive intersected so far.
	i32 hitInstanceId = IGNORE_ID;
	ei::Vec3 hitBarycentric;
	float hitT = tmax;						// t-value of the closest intersection.

	if(scene.numInstances == 1) {
		i32 traversalStack[OBJ_STACK_SIZE];
		first_intersection_scene_obj_lbvh(
			scene, ray, startInsPrimId, 0, traversalStack,
			hitT, hitInstanceId, hitPrimId, hitBarycentric);
	} else {
		const ei::Vec3 invDir = sdiv(1.0f, ray.direction);
		const ei::Vec3 ood = ray.origin * invDir;

		// Setup traversal.
		// Traversal stack in CUDA thread-local memory.
		i32 traversalStack[STACK_SIZE];
		traversalStack[0] = EntrypointSentinel;	// Bottom-most entry.
		i32 nodeAddr = 0; // Start from the root.
		i32 primCount = 0; // Internal nodes have no primitives
		i32* stackPtr = traversalStack; // Current position in traversal stack.

		// Traversal loop.
		while(nodeAddr != EntrypointSentinel) {
			if(nodeAddr < bvh.numInternalNodes) { // Internal node?
				// Fetch AABBs of the two child bvh.
				i32 nodeIdx = nodeAddr * 4;
				const ei::Vec4 Lmin_cL = bvh.bvh[nodeIdx];
				const ei::Vec4 Lmax_nL = bvh.bvh[nodeIdx + 1];
				const ei::Vec4 Rmin_cR = bvh.bvh[nodeIdx + 2];
				const ei::Vec4 Rmax_nR = bvh.bvh[nodeIdx + 3];

				// Intersect the ray against the children bounds.
				float c0min, c1min;
				bool traverseChild0 = intersect(ei::Vec3{Lmin_cL}, ei::Vec3{Lmax_nL}, invDir, ood, tmin, tmax, c0min);
				bool traverseChild1 = intersect(ei::Vec3{Rmin_cR}, ei::Vec3{Rmax_nR}, invDir, ood, tmin, tmax, c1min);

				// Neither child was intersected => pop stack.
				if(!traverseChild0 && !traverseChild1) {
					nodeAddr = *stackPtr;
					--stackPtr;
					if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
						primCount = *stackPtr;
						--stackPtr;
					}
				}
				// Otherwise => fetch child pointers.
				else {
					nodeAddr = traverseChild0 ? float_bits_as_int(Lmin_cL.w) : float_bits_as_int(Rmin_cR.w);
					primCount = traverseChild0 ? float_bits_as_int(Lmax_nL.w) : float_bits_as_int(Rmax_nR.w);

					// Both children were intersected => push the farther one.
					if (traverseChild0 && traverseChild1) {
						i32 pushAddr = float_bits_as_int(Rmin_cR.w); // nodeAddr is Lmin_cL.w, this is the other one
						i32 pushCount = float_bits_as_int(Rmax_nR.w);
						if (c1min < c0min) {
							i32 tmp = nodeAddr;
							nodeAddr = pushAddr;
							pushAddr = tmp;
							tmp = primCount;
							primCount = pushCount;
							pushCount = tmp;
						}
						if(pushAddr >= bvh.numInternalNodes) { // Leaf? Then push the count too
							++stackPtr;
							*stackPtr = pushCount;
						}
						++stackPtr;
						*stackPtr = pushAddr;
					}
				}
			}
			
			if(nodeAddr >= bvh.numInternalNodes && nodeAddr != EntrypointSentinel) { // Leaf?
				const i32 instanceId = bvh.primIds[ nodeAddr - bvh.numInternalNodes ];

				// TODO: no loop here! better use only one 'primitive' and wait for the next while iteration
				for(i32 i = 0; i < primCount; i++) {
					first_intersection_scene_obj_lbvh(scene, ray, startInsPrimId,
						instanceId, stackPtr+1, hitT, hitInstanceId, hitPrimId,
						hitBarycentric);
				}

				// Pop next node.
				nodeAddr = *stackPtr;
				--stackPtr;
				if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
					primCount = *stackPtr;
					--stackPtr;
				}
			}
		}
	}

	// Nobody should update hitT if no primitive is hit
	mAssert((hitInstanceId != IGNORE_ID && hitPrimId != IGNORE_ID) || hitT == tmax);

	/* TEST CODE WHICH MAKES A LINEAR TEST (without the BVH)
	for(int i = 0; i < scene.numInstances; ++i) {
		auto& obj = scene.objects[ scene.objectIndices[i] ];
		const ei::Mat3x3 invRotScale = ei::invert(ei::Mat3x3{scene.transformations[i]});
		const ei::Vec3 invTranslation { -scene.transformations[i][3],
										-scene.transformations[i][7],
										-scene.transformations[i][11] };
		ei::Ray transRay = { invRotScale * (ray.origin + invTranslation),
							 normalize(invRotScale * ray.direction) };
		for(int p = 0; p < obj.numPrimitives; ++p) {
			if(intersects_primitve(obj, transRay, p, -1, hitPrimId, hitT, hitBarycentric))
				hitInstanceId = i;
		}
	}*/

	if(hitInstanceId == IGNORE_ID) {
		return { hitT, { IGNORE_ID, IGNORE_ID } };
	} else {
		// To be determined
		ei::Vec3 normal;
		ei::Vec3 tangent;
		ei::Vec2 uv;

		bool hitSecondTri = false;
		i32 primId = hitPrimId;
		if (hitPrimId & SECOND_QUAD_TRIANGLE_BIT) {
			primId = hitPrimId & SECOND_QUAD_TRIANGLE_MASK;
			hitSecondTri = true;
		}

		const ObjectDescriptor<dev>& obj = scene.objects[ scene.objectIndices[hitInstanceId] ];
		const i32 offsetSpheres = obj.polygon.numTriangles + obj.polygon.numQuads;
		if(primId >= offsetSpheres) { // Sphere?
			const i32 sphId = primId - offsetSpheres;
			const ei::Vec3 hitPoint = ray.origin + hitT * ray.direction;
			normal = normalize(hitPoint - obj.spheres.spheres[sphId].center);

			// Normalization is done later
			if(normal.x == 0.f && normal.y == 0.f)
				tangent = ei::Vec3(1.f, 0.f, 0.f);
			else
				tangent = ei::Vec3(ei::Vec2(normal.y, -normal.x), 0.f);

			uv.x = atan2f(normal.x, normal.y) / (2.f * ei::PI) + 0.5f;
			uv.y = 0.5f * normal.z + 0.5f;
		} else {
			const i32* indices = (i32*)obj.polygon.vertexIndices;
			const ei::Vec3* meshVertices = obj.polygon.vertices;
			const ei::Vec2* meshUVs = obj.polygon.uvs;
			i32 triId;
			if(primId < (i32)obj.polygon.numTriangles) {
				// Triangle.
				triId = primId * 3;
			} else {
				// Quad.
				triId = (primId - obj.polygon.numTriangles) * 4;
				indices += obj.polygon.numTriangles * 3;
				// TODO: compute quad interpolation coordinate
			}
			ei::IVec3 ids = { indices[triId],
							  hitSecondTri ? indices[triId + 3] : indices[triId + 1],
							  indices[triId + 2] };

			ei::Vec3 v[3] = { meshVertices[ids.x], meshVertices[ids.y], meshVertices[ids.z] };
			tangent = v[1] - v[0]; // TODO: fetch the instance instead (issue #44)
			normal = ei::cross(v[0] - v[2], tangent);

			ei::Vec2 uvV[3] = { meshUVs[ids.x], meshUVs[ids.y], meshUVs[ids.z] };
			uv = uvV[0] * hitBarycentric.x + uvV[1] * hitBarycentric.y +
				 uvV[2] * hitBarycentric.z;
		}

		// TODO: enable this for (probably) better code?
		//normal = ei::normalize(ei::transformDir(normal, transforms[hitInstanceId]));
		//tangent = ei::normalize(ei::transformDir(tangent, transforms[hitInstanceId]));
		const ei::Mat3x3 transMatrix = ei::Mat3x3{ scene.transformations[hitInstanceId] };
		normal = ei::normalize(transMatrix * normal);
		tangent = ei::normalize(transMatrix * tangent);

		return { hitT, { hitInstanceId, hitPrimId }, normal, tangent, uv, hitBarycentric };
	}
}

template < Device dev > CUDA_FUNCTION
bool any_intersection_scene_obj_lbvh(
	const SceneDescriptor<dev>& scene,
	const ei::Ray ray,
	const RayIntersectionResult::HitID& startInsPrimId,
	const i32 instanceId,
	float tmax,
	i32* traversalStack
) {
	const ei::Mat3x3 invRotScale = ei::invert(ei::Mat3x3{scene.transformations[instanceId]});
	const ei::Vec3 invTranslation { -scene.transformations[instanceId][3],
									-scene.transformations[instanceId][7],
									-scene.transformations[instanceId][11] };
	ei::Ray transRay = { invRotScale * (ray.origin + invTranslation),
						 normalize(invRotScale * ray.direction) };
	const ei::Vec3 invDir = sdiv(1.0f, transRay.direction);
	const ei::Vec3 ood = transRay.origin * invDir;

	const i32 objId = scene.objectIndices[instanceId];
	const ei::Box& box = scene.aabbs[objId];
	const float tmin = 1e-6f * len(box.max - box.min);

	// Intersect the ray against the obj bounding box.
	float hitT;
	if(intersect(box.min, box.max, invDir, ood, tmin, tmax, hitT)) {
		// Intersect the ray against the obj primtive bvh.
		const ObjectDescriptor<dev>& obj = scene.objects[objId];
		const LBVH* lbvh = (LBVH*)obj.accelStruct.accelParameters;
		const i32 checkPrimId = (startInsPrimId.instanceId == instanceId) ? startInsPrimId.primId : IGNORE_ID;
		// Do ray-obj test.
		return any_intersection_obj_lbvh_imp(*lbvh, obj, transRay, checkPrimId,
			invDir, ood, tmin, tmax, traversalStack);
	}
	return false;
}

template < Device dev > CUDA_FUNCTION
bool any_intersection_scene_lbvh_imp(
	const LBVH& bvh,
	const SceneDescriptor<dev>& scene,
	const ei::Ray& ray,
	const RayIntersectionResult::HitID& startInsPrimId,
	const float tmax
) {
	const ei::Vec3 invDir = sdiv(1.0f, ray.direction);
	const ei::Vec3 ood = ray.origin * invDir;
	const float tmin = 1e-6f * len(scene.aabb.max - scene.aabb.min);

	if(scene.numInstances == 1) {
		i32 traversalStack[OBJ_STACK_SIZE];
		return any_intersection_scene_obj_lbvh(scene, ray, startInsPrimId,
			0, tmax, traversalStack);
	} else {
		// Setup traversal.
		// Traversal stack in CUDA thread-local memory.
		i32 traversalStack[STACK_SIZE];
		traversalStack[0] = EntrypointSentinel;	// Bottom-most entry.
		i32 nodeAddr = 0; // Start from the root.  
		i32* stackPtr = traversalStack; // Current position in traversal stack.
		i32 primCount = 0; // Internal nodes have no primitives

		// Traversal loop.
		while (nodeAddr != EntrypointSentinel) {
			if(nodeAddr < bvh.numInternalNodes) { // Internal node?
				// Fetch AABBs of the two child bvh.
				i32 nodeIdx = nodeAddr * 4;
				const ei::Vec4 Lmin_cL = bvh.bvh[nodeIdx];
				const ei::Vec4 Lmax_nL = bvh.bvh[nodeIdx + 1];
				const ei::Vec4 Rmin_cR = bvh.bvh[nodeIdx + 2];
				const ei::Vec4 Rmax_nR = bvh.bvh[nodeIdx + 3];

				// Intersect the ray against the child bvh.
				float c0min, c1min;
				bool traverseChild0 = intersect(ei::Vec3{Lmin_cL}, ei::Vec3{Lmax_nL}, invDir, ood, tmin, tmax, c0min);
				bool traverseChild1 = intersect(ei::Vec3{Rmin_cR}, ei::Vec3{Rmax_nR}, invDir, ood, tmin, tmax, c1min);

				// Neither child was intersected => pop stack.
				if (!traverseChild0 && !traverseChild1) {
					nodeAddr = *stackPtr;
					--stackPtr;
					if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
						primCount = *stackPtr;
						--stackPtr;
					}
				}
				// Otherwise => fetch child pointers.
				else {
					nodeAddr = traverseChild0 ? float_bits_as_int(Lmin_cL.w) : float_bits_as_int(Rmin_cR.w);
					primCount = traverseChild0 ? float_bits_as_int(Lmax_nL.w) : float_bits_as_int(Rmax_nR.w);

					// Both children were intersected => push the farther one.
					if (traverseChild0 && traverseChild1) {
						i32 pushAddr = float_bits_as_int(Rmin_cR.w); // nodeAddr is Lmin_cL.w, this is the other one
						i32 pushCount = float_bits_as_int(Rmax_nR.w);
						if (c1min < c0min) {
							i32 tmp = nodeAddr;
							nodeAddr = pushAddr;
							pushAddr = tmp;
							tmp = primCount;
							primCount = pushCount;
							pushCount = tmp;
						}
						if(pushAddr >= bvh.numInternalNodes) { // Leaf? Then push the count too
							++stackPtr;
							*stackPtr = pushCount;
						}
						++stackPtr;
						*stackPtr = pushAddr;
					}
				}
			}

			if(nodeAddr >= bvh.numInternalNodes && nodeAddr != EntrypointSentinel) { // Leaf?
				const i32 instanceId = bvh.primIds[ nodeAddr - bvh.numInternalNodes ];

				// TODO: no loop here! better use only one 'primitive' and wait for the next while iteration
				for(i32 i = 0; i < primCount; i++) {
					if(any_intersection_scene_obj_lbvh(scene, ray, startInsPrimId,
						instanceId, tmax, stackPtr+1))
						return true;
				}

				// Pop next node.
				nodeAddr = *stackPtr;
				--stackPtr;
				if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
					primCount = *stackPtr;
					--stackPtr;
				}
			}
		}
		return false;
	}
}

template < Device dev > CUDA_FUNCTION
bool any_intersection_scene_lbvh(
	const SceneDescriptor<dev>& scene,
	const ei::Ray& ray,
	const RayIntersectionResult::HitID& startInsPrimId,
	const float tmax
) {
	const LBVH* lbvh = (const LBVH*)scene.accelStruct.accelParameters;
	return any_intersection_scene_lbvh_imp<dev>(
		*lbvh, scene, ray, startInsPrimId, tmax);
}

template __host__ __device__ bool any_intersection_scene_lbvh(
	const SceneDescriptor<Device::CUDA>& scene,
	const ei::Ray& ray, const RayIntersectionResult::HitID& startInsPrimId,
	const float tmax
);

template __host__ __device__ bool any_intersection_scene_lbvh(
	const SceneDescriptor<Device::CPU>& scene,
	const ei::Ray& ray, const RayIntersectionResult::HitID& startInsPrimId,
	const float tmax
);

template < Device dev > CUDA_FUNCTION
RayIntersectionResult first_intersection_scene_lbvh(
	const SceneDescriptor<dev>& scene,
	const ei::Ray& ray,
	const RayIntersectionResult::HitID& startInsPrimId,
	const float tmax
) {
	const LBVH* lbvh = (const LBVH*)scene.accelStruct.accelParameters;
	return first_intersection_scene_lbvh_imp<dev>(
		*lbvh, scene, ray, startInsPrimId, tmax);
}

template __host__ __device__ RayIntersectionResult first_intersection_scene_lbvh(
	const SceneDescriptor<Device::CUDA>&,
	const ei::Ray&,
	const RayIntersectionResult::HitID&,
	const float
);

template __host__ __device__ RayIntersectionResult first_intersection_scene_lbvh(
	const SceneDescriptor<Device::CPU>& ,
	const ei::Ray&,
	const RayIntersectionResult::HitID&,
	const float
);

}}} // namespace mufflon::scene::accel_struct
