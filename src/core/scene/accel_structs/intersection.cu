#include "hip/hip_runtime.h"
#include "intersection.hpp"
#include "accel_structs_commons.hpp"
#include "lbvh.hpp"
#include "util/types.hpp"

#include <hip/hip_runtime_api.h>
#include <ei/3dtypes.hpp>
#include <ei/3dintersection.hpp>

namespace mufflon { namespace scene { namespace accel_struct {

namespace {

constexpr float SCENE_SCALE_EPS = 1e-4f;

CUDA_FUNCTION __forceinline void add_epsilon(ei::Ray& ray, const ei::Vec3& geoNormal) {
	ei::Vec3 offset = geoNormal * SCENE_SCALE_EPS;
	if(dot(geoNormal, ray.direction) >= 0.0f)
		ray.origin += offset;
	else
		ray.origin -= offset;
}


#define STACK_SIZE              96 //64          // Size of the traversal stack in local memory.
#define OBJ_STACK_SIZE              64 //64          // Size of the traversal stack in local memory.
enum : i32 {
	EntrypointSentinel = (i32)0xFFFFFFFF,   // Bottom-most stack entry, indicating the end of traversal.
	IGNORE_ID = (i32)0xFFFFFFFF
};

// Experimentally determined best mix of float/i32/video minmax instructions for Kepler.
__device__ __inline__ i32   min_min(i32 a, i32 b, i32 c) { i32 v; asm("vmin.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ i32   min_max(i32 a, i32 b, i32 c) { i32 v; asm("vmin.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ i32   max_min(i32 a, i32 b, i32 c) { i32 v; asm("vmax.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ i32   max_max(i32 a, i32 b, i32 c) { i32 v; asm("vmax.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ float fmin_fmin(float a, float b, float c) { return __int_as_float(min_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmin_fmax(float a, float b, float c) { return __int_as_float(min_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmin(float a, float b, float c) { return __int_as_float(max_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmax(float a, float b, float c) { return __int_as_float(max_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float spanBeginKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d) { return fmax_fmax(fminf(a0, a1), fminf(b0, b1), fmin_fmax(c0, c1, d)); }
__device__ __inline__ float spanEndKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d) { return fmin_fmin(fmaxf(a0, a1), fmaxf(b0, b1), fmax_fmin(c0, c1, d)); }


CUDA_FUNCTION bool intersect(const ei::Vec3& boxMin, const ei::Vec3& boxMax,
	const ei::Vec3 invDir, const ei::Vec3 ood, 
	const float tmin, const float tmax, float& cmin) {//, float& cmax) {
#ifdef __CUDA_ARCH__
	ei::Vec3 lo = boxMin * invDir - ood;
	ei::Vec3 hi = boxMax * invDir - ood;
	cmin = spanBeginKepler(lo.x, hi.x, lo.y, hi.y, lo.z, hi.z, tmin);
	const float cmax = spanEndKepler(lo.x, hi.x, lo.y, hi.y, lo.z, hi.z, tmax);
	return cmin <= cmax;
#else
	// TODO: use the epsilon one? FastRay one?
	float t0 = boxMin.x * invDir.x - ood.x;
	float t1 = boxMax.x * invDir.x - ood.x;
	cmin = ei::min(t0, t1);
	float cmax = ei::max(t0, t1);
	if (cmax < tmin || cmin > tmax) return false;
	t0 = boxMin.y * invDir.y - ood.y;
	t1 = boxMax.y * invDir.y - ood.y;
	float min2 = ei::min(t0, t1);
	float max2 = ei::max(t0, t1);
	cmin = ei::max(cmin, min2);
	cmax = ei::min(cmax, max2);
	if (cmax < tmin || cmin > tmax || cmin > cmax) return false;
	t0 = boxMin.z * invDir.z - ood.z;
	t1 = boxMax.z * invDir.z - ood.z;
	min2 = ei::min(t0, t1);
	max2 = ei::max(t0, t1);
	cmin = ei::max(cmin, min2);
	cmax = ei::min(cmax, max2);
	return (cmax >= tmin) && (cmin <= tmax) && (cmin <= cmax);
#endif // __CUDA_ARCH__
}

// Helper functions for the intersection test
CUDA_FUNCTION __forceinline__ float computeU(const float v, const float A1, const float A2,
											 const float B1, const float B2, const float C1,
											 const float C2, const float D1, const float D2) {
	const float a = v * A2 + B2;
	const float b = v * (A2 - A1) + B2 - B1;
	if(ei::abs(b) >= ei::abs(a))
		return (v*(C1 - C2) + D1 - D2) / b;
	else
		return (-v * C2 - D2) / a;
}

// Quad intersection test
CUDA_FUNCTION float intersectQuad(const ei::Tetrahedron& quad, const ei::Ray& ray, ei::Vec2& uv) {
	// Implementation from http://www.sci.utah.edu/~kpotter/publications/ramsey-2004-RBPI.pdf
	const ei::Vec3& p00 = quad.v0;
	const ei::Vec3& p10 = quad.v1;
	const ei::Vec3& p01 = quad.v3;
	const ei::Vec3& p11 = quad.v2;

	const ei::Vec3 a = p11 - p10 - p01 + p00;
	const ei::Vec3 b = p10 - p00;
	const ei::Vec3 c = p01 - p00;
	const ei::Vec3 d = p00;

	const float AXY = a.y*ray.direction.x - a.x*ray.direction.y;
	const float AXZ = a.z*ray.direction.x - a.x*ray.direction.z;
	const float AYZ = a.z*ray.direction.y - a.y*ray.direction.z;
	const float BXY = b.y*ray.direction.x - b.x*ray.direction.y;
	const float BXZ = b.z*ray.direction.x - b.x*ray.direction.z;
	const float BYZ = b.z*ray.direction.y - b.y*ray.direction.z;
	const float CXY = c.y*ray.direction.x - c.x*ray.direction.y;
	const float CXZ = c.z*ray.direction.x - c.x*ray.direction.z;
	const float CYZ = c.z*ray.direction.y - c.y*ray.direction.z;
	const float DXY = (d.y - ray.origin.y) * ray.direction.x - (d.x - ray.origin.x) * ray.direction.y;
	const float DXZ = (d.z - ray.origin.z) * ray.direction.x - (d.x - ray.origin.x) * ray.direction.z;
	const float DYZ = (d.z - ray.origin.z) * ray.direction.y - (d.y - ray.origin.y) * ray.direction.z;

	float A1, A2, B1, B2, C1, C2, D1, D2;
		// Use the component with largest ray direction component to avoid singularities
	if(ei::abs(ray.direction.x) >= ei::abs(ray.direction.y) && ei::abs(ray.direction.x) >= ei::abs(ray.direction.z)) {
		A1 = AXY;
		B1 = BXY;
		C1 = CXY;
		D1 = DXY;
		A2 = AXZ;
		B2 = BXZ;
		C2 = CXZ;
		D2 = DXZ;
	} else if(ei::abs(ray.direction.y) >= ei::abs(ray.direction.z)) {
		A1 = -AXY;
		B1 = -BXY;
		C1 = -CXY;
		D1 = -DXY;
		A2 = AYZ;
		B2 = BYZ;
		C2 = CYZ;
		D2 = DYZ;
	} else {
		A1 = AXZ;
		B1 = BXZ;
		C1 = CXZ;
		D1 = DXZ;
		A2 = AYZ;
		B2 = BYZ;
		C2 = CYZ;
		D2 = DYZ;
	}

	// Solve quadratic equ. for number of hitpoints
	float t = -1.f;
	float v0, v1;
	if(ei::solveSquarePoly(A2*C1 - A1*C2, A2*D1 - A1*D2 + B2*C1 - B1*C2, B2*D1 - B1*D2, v0, v1)) {
		// For the sake of divergence ignore the fact we might only have a single solution
		float u0, u1;
		float t0 = -1.f;
		float t1 = -1.f;
		if(v0 >= 0.f && v0 <= 1.f) {
			u0 = computeU(v0, A1, A2, B1, B2, C1, C2, D1, D2);
			if(u0 >= 0.f && u0 <= 1.f) {
				ei::Vec3 test = (u0*v0 * a + u0*b + v0*c + d - ray.origin) / ray.direction;
				if(ei::abs(ray.direction.x) >= ei::abs(ray.direction.y) &&
				   ei::abs(ray.direction.x) >= ei::abs(ray.direction.z))
					t0 = (u0*v0*a.x + u0 * b.x + v0 * c.x + d.x - ray.origin.x) / ray.direction.x;
				else if(ei::abs(ray.direction.y) >= ei::abs(ray.direction.z))
					t0 = (u0*v0*a.y + u0 * b.y + v0 * c.y + d.y - ray.origin.y) / ray.direction.y;
				else
					t0 = (u0*v0*a.z + u0 * b.z + v0 * c.z + d.z - ray.origin.z) / ray.direction.z;
			}
		}
		if(v1 >= 0.f && v1 <= 1.f) {
			u1 = computeU(v1, A1, A2, B1, B2, C1, C2, D1, D2);
			if(u1 >= 0.f && u1 <= 1.f) {
				ei::Vec3 test = (u1*v1 * a + u1*b + v1*c + d - ray.origin) / ray.direction;
				if(ei::abs(ray.direction.x) >= ei::abs(ray.direction.y) &&
				   ei::abs(ray.direction.x) >= ei::abs(ray.direction.z))
					t1 = (u1*v1*a.x + u1 * b.x + v1 * c.x + d.x - ray.origin.x) / ray.direction.x;
				else if(ei::abs(ray.direction.y) >= ei::abs(ray.direction.z))
					t1 = (u1*v1*a.y + u1 * b.y + v1 * c.y + d.y - ray.origin.y) / ray.direction.y;
				else
					t1 = (u1*v1*a.z + u1 * b.z + v1 * c.z + d.z - ray.origin.z) / ray.direction.z;
			}
		}
		if(t0 > 0.f) {
			if(t1 > 0.f && t1 < t0) {
				uv = ei::Vec2(u1, v1);
				t = t1;
			} else {
				uv = ei::Vec2(u0, v0);
				t = t0;
			}
		} else if(t1 > 0.f) {
			uv = ei::Vec2(u1, v1);
			t = t1;
		}
	}

	return t;
}

template < Device dev >
CUDA_FUNCTION bool intersects_primitve(
	const LodDescriptor<dev>& obj,
	const ei::Ray& ray,
	const i32 primId,
	int& hitPrimId,
	float& hitT,				// In out: max hit distance before, if hit then returns the new distance
	SurfaceParametrization& surfParams
) {
	if(primId < (i32)obj.polygon.numTriangles) {
		// Triangle.
		const ei::Vec3* meshVertices = obj.polygon.vertices;
		const i32 indexOffset = primId * 3;
		const ei::IVec3 ids = { obj.polygon.vertexIndices[indexOffset],
								obj.polygon.vertexIndices[indexOffset + 1],
								obj.polygon.vertexIndices[indexOffset + 2] };
		const ei::Triangle tri = { meshVertices[ids[0]],
								   meshVertices[ids[1]],
								   meshVertices[ids[2]] };

		float t;
		ei::Vec3 barycentric;
		if(ei::intersects(ray, tri, t, barycentric) && t < hitT && t > 0.0f) {
			hitT = t;
			surfParams.barycentric = ei::Vec2{ barycentric.x, barycentric.y };
			hitPrimId = primId;
			return true;
		}
	} else if(primId < (i32)(obj.polygon.numTriangles + obj.polygon.numQuads)) {
		// Quad.
		const i32 indexOffset = (primId - obj.polygon.numTriangles) * 4 + obj.polygon.numTriangles * 3;
		const ei::Vec3* meshVertices = obj.polygon.vertices;

		// if(startPrimId == primId) return false; // TODO: #78 Masking to avoid self intersections
		const ei::IVec4 ids = { obj.polygon.vertexIndices[indexOffset],
								obj.polygon.vertexIndices[indexOffset + 1],
								obj.polygon.vertexIndices[indexOffset + 2],
								obj.polygon.vertexIndices[indexOffset + 3] };
		const ei::Tetrahedron quad = { meshVertices[ids[0]],
										meshVertices[ids[1]],
										meshVertices[ids[2]],
										meshVertices[ids[3]] };
		ei::Vec2 bilinear;
		// There are up to two intersections with a quad. Since the closer one
		// could be the self intersection move forward on the ray before testing.
		const float t = intersectQuad(quad, ray, bilinear);

		if(t > 0.0f && t < hitT) {
			hitT = t;
			surfParams.bilinear = bilinear;
			hitPrimId = primId;
			return true;
		}
	} else {
		// Sphere.
		// Masking not possible for spheres: in case of transparent objects we need
		// self intersections inside.
		const ei::Sphere& sph = obj.spheres.spheres[primId];
		// Because it is important if we start incide or outside it is better
		// to modify the ray beforehand. Testing for tmin afterwards is buggy.
		float t;
		// TODO: use some epsilon?
		if(ei::intersects(ray, sph, t) && t < hitT) {
			hitT = t;
			hitPrimId = primId;
			// Barycentrics unused; TODO: get coordinates anyway?
			return true;
		}
	}
	return false;
}

} // namespace ::

template < Device dev >
CUDA_FUNCTION bool any_intersection_obj_lbvh_imp(
	const LBVH& bvh,
	const LodDescriptor<dev>& obj,
	const ei::Ray& ray,
	const ei::Vec3& invDir, 
	const ei::Vec3& ood,
	const float tmax,
	i32* traversalStack
) {
	// Since all threads go to the following branch if numPrimitives == 1,
	// there is no problem with branching.
	if(obj.numPrimitives == 1) {
		float hitT = tmax;
		SurfaceParametrization surfParams;
		i32 hitPrimitiveId;
		if(intersects_primitve(obj, ray, 0, hitPrimitiveId, hitT, surfParams)) {
			return true;
		}
		return false;
	}

	// Setup traversal.
	traversalStack[0] = EntrypointSentinel;	// Bottom-most entry.
	i32 nodeAddr = 0; // Start from the root.  
	i32* stackPtr = traversalStack; // Current position in traversal stack.
	i32 primCount = 0; // Internal nodes have no primitives

	// Traversal loop.
	while(nodeAddr != EntrypointSentinel) {
		if(nodeAddr < bvh.numInternalNodes) { // Internal node?
			// Fetch AABBs of the two child bvh.
			i32 nodeIdx = nodeAddr * 4;
			const ei::Vec4 Lmin_cL = bvh.bvh[nodeIdx];
			const ei::Vec4 Lmax_nL = bvh.bvh[nodeIdx + 1];
			const ei::Vec4 Rmin_cR = bvh.bvh[nodeIdx + 2];
			const ei::Vec4 Rmax_nR = bvh.bvh[nodeIdx + 3];

			// Intersect the ray against the children bounds.
			float c0min, c1min;
			bool traverseChild0 = intersect(ei::Vec3{Lmin_cL}, ei::Vec3{Lmax_nL}, invDir, ood, 0.0f, tmax, c0min);
			bool traverseChild1 = intersect(ei::Vec3{Rmin_cR}, ei::Vec3{Rmax_nR}, invDir, ood, 0.0f, tmax, c1min);

			// Neither child was intersected => pop stack.
			if(!traverseChild0 && !traverseChild1) {
				nodeAddr = *stackPtr;
				--stackPtr;
				if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
					primCount = *stackPtr;
					--stackPtr;
				}
			}
			// Otherwise => fetch child pointers.
			else {
				nodeAddr = traverseChild0 ? float_bits_as_int(Lmin_cL.w) : float_bits_as_int(Rmin_cR.w);
				primCount = traverseChild0 ? float_bits_as_int(Lmax_nL.w) : float_bits_as_int(Rmax_nR.w);

				// Both children were intersected => push the farther one.
				if (traverseChild0 && traverseChild1) {
					i32 pushAddr = float_bits_as_int(Rmin_cR.w); // nodeAddr is Lmin_cL.w, this is the other one
					i32 pushCount = float_bits_as_int(Rmax_nR.w);
					if (c1min < c0min) {
						i32 tmp = nodeAddr;
						nodeAddr = pushAddr;
						pushAddr = tmp;
						tmp = primCount;
						primCount = pushCount;
						pushCount = tmp;
					}
					if(pushAddr >= bvh.numInternalNodes) { // Leaf? Then push the count too
						++stackPtr;
						*stackPtr = pushCount;
					}
					++stackPtr;
					*stackPtr = pushAddr;
				}
			}
		}

		if(nodeAddr >= bvh.numInternalNodes && nodeAddr != EntrypointSentinel) { // Leaf?
			const i32 primId = nodeAddr - bvh.numInternalNodes;

			// TODO: no loop here! better use only one 'primitive' and wait for the next while iteration
			for(i32 i = 0; i < primCount; i++) {
				float hitT = tmax;
				SurfaceParametrization surfParams;
				i32 hitPrimitiveId;
				if(intersects_primitve(obj, ray, bvh.primIds[primId + i], hitPrimitiveId, hitT, surfParams))
					return true;
			}

			// Pop next node.
			nodeAddr = *stackPtr;
			--stackPtr;
			if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
				primCount = *stackPtr;
				--stackPtr;
			}
		}
	}
	return false;
}


template < Device dev >
CUDA_FUNCTION bool first_intersection_obj_lbvh_imp(
	const LBVH& bvh,
	const LodDescriptor<dev>& obj,
	const ei::Ray& ray,
	const ei::Vec3& invDir, 
	const ei::Vec3& ood,
	int& hitPrimId, float& hitT,
	SurfaceParametrization& surfParams,
	i32* traversalStack
) {
	// Fast path - no BVH
	if(obj.numPrimitives == 1) {
		return intersects_primitve(obj, ray, 0,
			hitPrimId, hitT, surfParams);
	}
	
	// Setup traversal.
	traversalStack[0] = EntrypointSentinel;	// Bottom-most entry.
	i32 nodeAddr = 0; // Start from the root.  
	i32* stackPtr = traversalStack; // Current position in traversal stack.
	i32 primCount = 0; // Internal nodes have no primitives

	bool hasHit = false;

	// Traversal loop.
	while(nodeAddr != EntrypointSentinel) {
		if(nodeAddr < bvh.numInternalNodes) { // Internal node?
			// Fetch AABBs of the two child bvh.
			i32 nodeIdx = nodeAddr * 4;
			const ei::Vec4 Lmin_cL = bvh.bvh[nodeIdx];
			const ei::Vec4 Lmax_nL = bvh.bvh[nodeIdx + 1];
			const ei::Vec4 Rmin_cR = bvh.bvh[nodeIdx + 2];
			const ei::Vec4 Rmax_nR = bvh.bvh[nodeIdx + 3];

			// Intersect the ray against the children bounds.
			float c0min, c1min;
			bool traverseChild0 = intersect(ei::Vec3{Lmin_cL}, ei::Vec3{Lmax_nL}, invDir, ood, 0.0f, hitT, c0min);
			bool traverseChild1 = intersect(ei::Vec3{Rmin_cR}, ei::Vec3{Rmax_nR}, invDir, ood, 0.0f, hitT, c1min);

			// Neither child was intersected => pop stack.
			if(!traverseChild0 && !traverseChild1) {
				nodeAddr = *stackPtr;
				--stackPtr;
				if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
					primCount = *stackPtr;
					--stackPtr;
				}
			}
			// Otherwise => fetch child pointers.
			else {
				nodeAddr = traverseChild0 ? float_bits_as_int(Lmin_cL.w) : float_bits_as_int(Rmin_cR.w);
				primCount = traverseChild0 ? float_bits_as_int(Lmax_nL.w) : float_bits_as_int(Rmax_nR.w);

				// Both children were intersected => push the farther one.
				if (traverseChild0 && traverseChild1) {
					i32 pushAddr = float_bits_as_int(Rmin_cR.w); // nodeAddr is Lmin_cL.w, this is the other one
					i32 pushCount = float_bits_as_int(Rmax_nR.w);
					if (c1min < c0min) {
						i32 tmp = nodeAddr;
						nodeAddr = pushAddr;
						pushAddr = tmp;
						tmp = primCount;
						primCount = pushCount;
						pushCount = tmp;
					}
					if(pushAddr >= bvh.numInternalNodes) { // Leaf? Then push the count too
						++stackPtr;
						*stackPtr = pushCount;
					}
					++stackPtr;
					*stackPtr = pushAddr;
				}
			}
		}

		if(nodeAddr >= bvh.numInternalNodes && nodeAddr != EntrypointSentinel) { // Leaf?
			const i32 primId = nodeAddr - bvh.numInternalNodes;

			// All intersection distances are in this instance's object space
			// TODO: no loop here! better use only one 'primitive' and wait for the next while iteration
			for(i32 i = 0; i < primCount; i++) {
				if(intersects_primitve(obj, ray, bvh.primIds[primId+i], hitPrimId, hitT, surfParams))
					hasHit = true;
			}

			// Pop next node.
			nodeAddr = *stackPtr;
			--stackPtr;
			if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
				primCount = *stackPtr;
				--stackPtr;
			}
		}
	}
	return hasHit;
}


template < Device dev > CUDA_FUNCTION
void first_intersection_scene_obj_lbvh(
	const SceneDescriptor<dev>& scene,
	const ei::Ray& ray,
	const i32 instanceId,
	i32* traversalStack,
	float& hitT,
	i32& hitInstanceId,
	i32& hitPrimId,
	SurfaceParametrization& surfParams
) {
	const ei::Vec3& invScale = 1.0f / scene.scales[instanceId];
	const ei::Mat3x3 invRot = ei::transpose(ei::Mat3x3{ scene.transformations[instanceId] });
	const ei::Vec3 invTranslation{ -scene.transformations[instanceId][3],
									-scene.transformations[instanceId][7],
									-scene.transformations[instanceId][11] };
	const ei::Vec3 rayDir = invScale * (invRot * ray.direction);
	const float rayScale = ei::len(rayDir);
	const float invRayScale = 1.f / rayScale;
	const ei::Ray transRay = { invScale * (invRot * (ray.origin + invTranslation)),
							   invRayScale * rayDir };
	const ei::Vec3 invDir = sdiv(1.0f, transRay.direction);
	const ei::Vec3 ood = transRay.origin * invDir;

	const i32 objId = scene.lodIndices[instanceId];
	const ei::Box& box = scene.aabbs[objId];

	// Scale our current maximum intersection distance into the object space to avoid false negatives
	float objSpaceHitT = hitT * rayScale;

	// Intersect the ray against the obj bounding box.
	float objSpaceT;
	if(intersect(box.min, box.max, invDir, ood, 0.0f, objSpaceHitT, objSpaceT)) {
		// Intersect the ray against the obj primitive bvh.
		const LodDescriptor<dev>& obj = scene.lods[objId];
		const LBVH* lbvh = (LBVH*)obj.accelStruct.accelParameters;
		if (first_intersection_obj_lbvh_imp(
			*lbvh, obj, transRay, invDir, ood, hitPrimId,
			objSpaceHitT, surfParams, traversalStack)) {
			// Translate the object-space distance into world space again
			hitT = invRayScale * objSpaceHitT;
			hitInstanceId = instanceId;
		}
	}
}

template < Device dev > __host__ __device__
RayIntersectionResult first_intersection(
	const SceneDescriptor<dev>& scene,
	ei::Ray& ray,
	const ei::Vec3& geoNormal,
	const float tmax
) {
	add_epsilon(ray, geoNormal);
	const LBVH& bvh = *(const LBVH*)scene.accelStruct.accelParameters;
	i32 hitPrimId = IGNORE_ID;						// No primitive intersected so far.
	i32 hitInstanceId = IGNORE_ID;
	SurfaceParametrization surfParams;
	float hitT = tmax;						// t-value of the closest intersection.

	if(scene.numInstances == 1) {
		i32 traversalStack[OBJ_STACK_SIZE];
		first_intersection_scene_obj_lbvh(
			scene, ray, 0, traversalStack,
			hitT, hitInstanceId, hitPrimId, surfParams);
	} else {
		const ei::Vec3 invDir = sdiv(1.0f, ray.direction);
		const ei::Vec3 ood = ray.origin * invDir;

		// Setup traversal.
		// Traversal stack in CUDA thread-local memory.
		i32 traversalStack[STACK_SIZE];
		traversalStack[0] = EntrypointSentinel;	// Bottom-most entry.
		i32 nodeAddr = 0; // Start from the root.
		i32 primCount = 0; // Internal nodes have no primitives
		i32* stackPtr = traversalStack; // Current position in traversal stack.

		// Traversal loop.
		while(nodeAddr != EntrypointSentinel) {
			if(nodeAddr < bvh.numInternalNodes) { // Internal node?
				// Fetch AABBs of the two child bvh.
				i32 nodeIdx = nodeAddr * 4;
				const ei::Vec4 Lmin_cL = bvh.bvh[nodeIdx];
				const ei::Vec4 Lmax_nL = bvh.bvh[nodeIdx + 1];
				const ei::Vec4 Rmin_cR = bvh.bvh[nodeIdx + 2];
				const ei::Vec4 Rmax_nR = bvh.bvh[nodeIdx + 3];

				// Intersect the ray against the children bounds.
				float c0min, c1min;
				bool traverseChild0 = intersect(ei::Vec3{Lmin_cL}, ei::Vec3{Lmax_nL}, invDir, ood, 0.0f, tmax, c0min);
				bool traverseChild1 = intersect(ei::Vec3{Rmin_cR}, ei::Vec3{Rmax_nR}, invDir, ood, 0.0f, tmax, c1min);

				// Neither child was intersected => pop stack.
				if(!traverseChild0 && !traverseChild1) {
					nodeAddr = *stackPtr;
					--stackPtr;
					if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
						primCount = *stackPtr;
						--stackPtr;
					}
				}
				// Otherwise => fetch child pointers.
				else {
					nodeAddr = traverseChild0 ? float_bits_as_int(Lmin_cL.w) : float_bits_as_int(Rmin_cR.w);
					primCount = traverseChild0 ? float_bits_as_int(Lmax_nL.w) : float_bits_as_int(Rmax_nR.w);

					// Both children were intersected => push the farther one.
					if (traverseChild0 && traverseChild1) {
						i32 pushAddr = float_bits_as_int(Rmin_cR.w); // nodeAddr is Lmin_cL.w, this is the other one
						i32 pushCount = float_bits_as_int(Rmax_nR.w);
						if (c1min < c0min) {
							i32 tmp = nodeAddr;
							nodeAddr = pushAddr;
							pushAddr = tmp;
							tmp = primCount;
							primCount = pushCount;
							pushCount = tmp;
						}
						if(pushAddr >= bvh.numInternalNodes) { // Leaf? Then push the count too
							++stackPtr;
							*stackPtr = pushCount;
						}
						++stackPtr;
						*stackPtr = pushAddr;
					}
				}
			}
			
			if(nodeAddr >= bvh.numInternalNodes && nodeAddr != EntrypointSentinel) { // Leaf?
				const i32 instanceId = nodeAddr - bvh.numInternalNodes;

				// TODO: no loop here! better use only one 'primitive' and wait for the next while iteration
				for(i32 i = 0; i < primCount; i++) {
					first_intersection_scene_obj_lbvh(scene, ray,
						bvh.primIds[ instanceId + i], stackPtr+1,
						hitT, hitInstanceId, hitPrimId, surfParams);
				}

				// Pop next node.
				nodeAddr = *stackPtr;
				--stackPtr;
				if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
					primCount = *stackPtr;
					--stackPtr;
				}
			}
		}
	}

	// Nobody should update hitT if no primitive is hit
	mAssert((hitInstanceId != IGNORE_ID && hitPrimId != IGNORE_ID) || hitT == tmax);

	/* TEST CODE WHICH MAKES A LINEAR TEST (without the BVH)
	for(int i = 0; i < scene.numInstances; ++i) {
		auto& obj = scene.lods[ scene.lodIndices[i] ];
		const ei::Mat3x3 invRotScale = ei::invert(ei::Mat3x3{scene.transformations[i]});
		const ei::Vec3 invTranslation { -scene.transformations[i][3],
										-scene.transformations[i][7],
										-scene.transformations[i][11] };
		ei::Ray transRay = { invRotScale * (ray.origin + invTranslation),
							 normalize(invRotScale * ray.direction) };
		for(int p = 0; p < obj.numPrimitives; ++p) {
			if(intersects_primitve(obj, transRay, p, -1, hitPrimId, hitT, surfParams))
				hitInstanceId = i;
		}
	}*/

	if(hitInstanceId == IGNORE_ID) {
		return { hitT, { IGNORE_ID, IGNORE_ID } };
	} else {
		// To be determined
		ei::Vec3 geoNormal;
		ei::Vec3 tangentX;
		ei::Vec3 tangentY;
		ei::Vec2 uv;

		i32 primId = hitPrimId;

		const LodDescriptor<dev>& obj = scene.lods[scene.lodIndices[hitInstanceId]];
		const ei::Vec3& scale = scene.scales[hitInstanceId];
		ei::Mat3x3 rotation = ei::Mat3x3{ scene.transformations[hitInstanceId] };

		const i32 offsetSpheres = obj.polygon.numTriangles + obj.polygon.numQuads;
		if(primId >= offsetSpheres) { // Sphere?
			const i32 sphId = primId - offsetSpheres;
			const ei::Vec3 hitPoint = ray.origin + hitT * ray.direction;
			const Point center { scene.transformations[hitInstanceId] * ei::Vec4(obj.spheres.spheres[sphId].center, 1.0f) };
			geoNormal = normalize(hitPoint - center); // Normalization required for acos() below

			if(geoNormal.x == 0.0f && geoNormal.y == 0.0f)
				tangentX = ei::Vec3(1.0f, 0.0f, 0.0f);
			else
				tangentX = ei::Vec3(normalize(ei::Vec2(geoNormal.y, -geoNormal.x)), 0.0f);
			tangentY = cross(geoNormal, tangentX);

			const ei::Vec3 localN = transpose(rotation) * geoNormal;
			uv.x = atan2f(localN.y, localN.x) / (2.0f * ei::PI) + 0.5f;
			uv.y = acosf(-localN.z) / ei::PI;
			surfParams.st = uv;
			return RayIntersectionResult{ hitT, { hitInstanceId, hitPrimId }, geoNormal, tangentX, tangentY, uv, surfParams };
		} else {
			const i32* indices = (i32*)obj.polygon.vertexIndices;
			const ei::Vec3* meshVertices = obj.polygon.vertices;
			const ei::Vec2* meshUVs = obj.polygon.uvs;
			if(primId < (i32)obj.polygon.numTriangles) {
				// Triangle.
				u32 triId = primId * 3;
				ei::IVec3 ids = { indices[triId],
								  indices[triId + 1],
								  indices[triId + 2] };
				const ei::Vec3 v[3] = { meshVertices[ids.x], meshVertices[ids.y], meshVertices[ids.z] };
				const ei::Vec2 uvV[3] = { meshUVs[ids.x], meshUVs[ids.y], meshUVs[ids.z] };
				// Compute the tangent space by solving LES
				const ei::Vec3 dx0 = v[1u] - v[0u];
				const ei::Vec3 dx1 = v[2u] - v[0u];
				const ei::Vec2 du0 = uvV[1u] - uvV[0u];
				const ei::Vec2 du1 = uvV[2u] - uvV[0u];
				float det = 1.f / (du0.x * du1.y - du0.y * du1.x);
				// TODO: fetch the instance instead (issue #44)
				// TODO: do the tangent's really need to be normalized?
				tangentX = det * (dx0 * du1.y - dx1 * du0.y);
				tangentY = det * (dx1 * du0.x - dx0 * du1.x);

				// Don't use the UV tangents to compute the normal, since they may be reversed
				geoNormal = cross(dx0, dx1);

				mAssert(dot(geoNormal, obj.polygon.normals[ids.x]) > 0.f);

				uv = uvV[0] * surfParams.barycentric.x + uvV[1] * surfParams.barycentric.y +
					uvV[2] * (1.f - surfParams.barycentric.x - surfParams.barycentric.y);
			} else {
				// Quad.
				i32 quadId = (primId - obj.polygon.numTriangles) * 4;
				indices += obj.polygon.numTriangles * 3;
				ei::IVec4 ids = { indices[quadId + 0],
								  indices[quadId + 1],
								  indices[quadId + 2],
								  indices[quadId + 3] };
				const ei::Vec3 v[4] = { meshVertices[ids.x], meshVertices[ids.y], meshVertices[ids.z], meshVertices[ids.w] };
				const ei::Vec2 uvV[4] = { meshUVs[ids.x], meshUVs[ids.y], meshUVs[ids.z], meshUVs[ids.w] };
				// Compute tangent space by using surrogate coordinate system to get interpolated UVs
				// https://stackoverflow.com/questions/47187600/differences-in-calculating-tbn-matrix-for-triangles-versus-quads
				// TODO: fetch the instance instead (issue #44)
				const ei::Vec3 dxds = (1.f - surfParams.bilinear.u) * (v[3u] - v[0u]) + surfParams.bilinear.u * (v[2u] - v[1u]);
				const ei::Vec3 dxdt = (1.f - surfParams.bilinear.v) * (v[1u] - v[0u]) + surfParams.bilinear.v * (v[2u] - v[3u]);
				const ei::Matrix<float, 3, 2> dxdst{
					dxds.x, dxdt.x,
					dxds.y, dxdt.y,
					dxds.z, dxdt.z
				};
				const ei::Vec2 duds = (1.f - surfParams.bilinear.u) * (uvV[3u] - uvV[0u]) + surfParams.bilinear.u * (uvV[2u] - uvV[1u]);
				const ei::Vec2 dudt = (1.f - surfParams.bilinear.v) * (uvV[1u] - uvV[0u]) + surfParams.bilinear.v * (uvV[2u] - uvV[3u]);
				const ei::Matrix<float, 2, 2> dudst{
					duds.x, dudt.x,
					duds.y, dudt.y,
				};
				const ei::Mat2x2 dsduv = ei::invert(dudst);
				const ei::Matrix<float, 3, 2> tangents = dxdst * dsduv;
				tangentX = ei::Vec3{ tangents(0, 0), tangents(1, 0), tangents(2, 0) };
				tangentY = ei::Vec3{ tangents(0, 1), tangents(1, 1), tangents(2, 1) };

				geoNormal = cross(dxdt, dxds);
				uv = ei::bilerp(uvV[0u], uvV[1u], uvV[3u], uvV[2u], surfParams.bilinear.x, surfParams.bilinear.y);
			}
		}

		// Transform the normal and tangents into world space
		// Polygon objects are allowed to have a non-uniform scaling
		geoNormal = normalize(rotation * (geoNormal / scale));
		tangentX = normalize(rotation * (tangentX / scale));
		tangentY = normalize(rotation * (tangentY / scale));

		mAssert(!(isnan(tangentX.x) || isnan(tangentX.y) || isnan(tangentX.z)));
		mAssert(!(isnan(tangentY.x) || isnan(tangentY.y) || isnan(tangentY.z)));
		mAssert(!(isnan(geoNormal.x) || isnan(geoNormal.y) || isnan(geoNormal.z)));

		return RayIntersectionResult{ hitT, { hitInstanceId, hitPrimId }, geoNormal, tangentX, tangentY, uv, surfParams };
	}
}

template < Device dev > __host__ __device__
bool any_intersection_scene_obj_lbvh(
	const SceneDescriptor<dev>& scene,
	const ei::Ray ray,
	const i32 instanceId,
	float tmax,
	i32* traversalStack
) {
	const ei::Vec3& invScale = 1.0f / scene.scales[instanceId];
	const ei::Mat3x3 invRot = ei::transpose(ei::Mat3x3{ scene.transformations[instanceId] });
	const ei::Vec3 invTranslation{ -scene.transformations[instanceId][3],
									-scene.transformations[instanceId][7],
									-scene.transformations[instanceId][11] };
	const ei::Vec3 rayDir = invScale * (invRot * ray.direction);
	const float rayScale = ei::len(rayDir);
	const float invRayScale = 1.f / rayScale;
	const ei::Ray transRay = { invScale * (invRot * (ray.origin + invTranslation)),
							   invRayScale * rayDir };
	const ei::Vec3 invDir = sdiv(1.0f, transRay.direction);
	const ei::Vec3 ood = transRay.origin * invDir;

	const i32 objId = scene.lodIndices[instanceId];
	const ei::Box& box = scene.aabbs[objId];

	// Scale our current maximum intersection distance into the object space to avoid false negatives
	const float objSpaceMaxT = tmax * rayScale;

	// Intersect the ray against the obj bounding box.
	float hitT;
	if(intersect(box.min, box.max, invDir, ood, 0.0f, objSpaceMaxT, hitT)) {
		// Intersect the ray against the obj primtive bvh.
		const LodDescriptor<dev>& obj = scene.lods[objId];
		const LBVH* lbvh = (LBVH*)obj.accelStruct.accelParameters;
		// Do ray-obj test.
		return any_intersection_obj_lbvh_imp(*lbvh, obj, transRay,
			invDir, ood, objSpaceMaxT, traversalStack);
	}
	return false;
}

template < Device dev > __host__ __device__
bool any_intersection(
	const SceneDescriptor<dev>& scene,
	ei::Ray ray,
	const ei::Vec3& geoNormal,
	const float maxDist
) {
	add_epsilon(ray, geoNormal);
	const LBVH& bvh = *(const LBVH*)scene.accelStruct.accelParameters;
	const ei::Vec3 invDir = sdiv(1.0f, ray.direction);
	const ei::Vec3 ood = ray.origin * invDir;
	const float tmax = maxDist - SCENE_SCALE_EPS * 2.0f; // Do not intersect the target surface

	if(scene.numInstances == 1) {
		i32 traversalStack[OBJ_STACK_SIZE];
		return any_intersection_scene_obj_lbvh(scene, ray,
			0, tmax, traversalStack);
	} else {
		// Setup traversal.
		// Traversal stack in CUDA thread-local memory.
		i32 traversalStack[STACK_SIZE];
		traversalStack[0] = EntrypointSentinel;	// Bottom-most entry.
		i32 nodeAddr = 0; // Start from the root.  
		i32* stackPtr = traversalStack; // Current position in traversal stack.
		i32 primCount = 0; // Internal nodes have no primitives

		// Traversal loop.
		while (nodeAddr != EntrypointSentinel) {
			if(nodeAddr < bvh.numInternalNodes) { // Internal node?
				// Fetch AABBs of the two child bvh.
				i32 nodeIdx = nodeAddr * 4;
				const ei::Vec4 Lmin_cL = bvh.bvh[nodeIdx];
				const ei::Vec4 Lmax_nL = bvh.bvh[nodeIdx + 1];
				const ei::Vec4 Rmin_cR = bvh.bvh[nodeIdx + 2];
				const ei::Vec4 Rmax_nR = bvh.bvh[nodeIdx + 3];

				// Intersect the ray against the child bvh.
				float c0min, c1min;
				bool traverseChild0 = intersect(ei::Vec3{Lmin_cL}, ei::Vec3{Lmax_nL}, invDir, ood, 0.0f, tmax, c0min);
				bool traverseChild1 = intersect(ei::Vec3{Rmin_cR}, ei::Vec3{Rmax_nR}, invDir, ood, 0.0f, tmax, c1min);

				// Neither child was intersected => pop stack.
				if (!traverseChild0 && !traverseChild1) {
					nodeAddr = *stackPtr;
					--stackPtr;
					if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
						primCount = *stackPtr;
						--stackPtr;
					}
				}
				// Otherwise => fetch child pointers.
				else {
					nodeAddr = traverseChild0 ? float_bits_as_int(Lmin_cL.w) : float_bits_as_int(Rmin_cR.w);
					primCount = traverseChild0 ? float_bits_as_int(Lmax_nL.w) : float_bits_as_int(Rmax_nR.w);

					// Both children were intersected => push the farther one.
					if (traverseChild0 && traverseChild1) {
						i32 pushAddr = float_bits_as_int(Rmin_cR.w); // nodeAddr is Lmin_cL.w, this is the other one
						i32 pushCount = float_bits_as_int(Rmax_nR.w);
						if (c1min < c0min) {
							i32 tmp = nodeAddr;
							nodeAddr = pushAddr;
							pushAddr = tmp;
							tmp = primCount;
							primCount = pushCount;
							pushCount = tmp;
						}
						if(pushAddr >= bvh.numInternalNodes) { // Leaf? Then push the count too
							++stackPtr;
							*stackPtr = pushCount;
						}
						++stackPtr;
						*stackPtr = pushAddr;
					}
				}
			}

			if(nodeAddr >= bvh.numInternalNodes && nodeAddr != EntrypointSentinel) { // Leaf?
				const i32 instanceId = nodeAddr - bvh.numInternalNodes;

				// TODO: no loop here! better use only one 'primitive' and wait for the next while iteration
				for(i32 i = 0; i < primCount; i++) {
					if(any_intersection_scene_obj_lbvh(scene, ray,
						bvh.primIds[ instanceId + i ], tmax, stackPtr+1))
						return true;
				}

				// Pop next node.
				nodeAddr = *stackPtr;
				--stackPtr;
				if(nodeAddr >= bvh.numInternalNodes) { // Leafs additionally store the primitive count
					primCount = *stackPtr;
					--stackPtr;
				}
			}
		}
		return false;
	}
}


template __host__ __device__ bool any_intersection(
	const SceneDescriptor<Device::CUDA>&,
	ei::Ray, const ei::Vec3&, const float
);

template __host__ __device__ bool any_intersection(
	const SceneDescriptor<Device::CPU>&,
	ei::Ray, const ei::Vec3&, const float
);

template __host__ __device__ RayIntersectionResult first_intersection(
	const SceneDescriptor<Device::CUDA>&, ei::Ray&, 
	const ei::Vec3&, const float
);

template __host__ __device__ RayIntersectionResult first_intersection(
	const SceneDescriptor<Device::CPU>&, ei::Ray&,
	const ei::Vec3&, const float
);

}}} // namespace mufflon::scene::accel_struct
