
#include <hip/hip_runtime.h>
#if 0
#include "silhouette_importance_gathering_pt.hpp"
#include "silhouette_pt_common.hpp"
#include "silhouette_pt_params.hpp"
#include <cuda_runtime.h>
#include <device_launch_parameters.h>

namespace mufflon { namespace renderer { namespace decimaters { namespace silhouette { namespace pt {

__global__ static void silhouette_kernel(pt::SilhouetteTargets::RenderBufferType<Device::CUDA> outputBuffer,
										 scene::SceneDescriptor<Device::CUDA>* scene,
										 const u32* seeds, SilhouetteParameters params,
										 Importances<Device::CUDA>** importances,
										 DeviceImportanceSums<Device::CUDA>* sums) {
	Pixel coord{
		threadIdx.x + blockDim.x * blockIdx.x,
		threadIdx.y + blockDim.y * blockIdx.y
	};
	if(coord.x >= outputBuffer.get_width() || coord.y >= outputBuffer.get_height())
		return;

	const int pixel = coord.x + coord.y * outputBuffer.get_width();

	math::Rng rng(seeds[pixel]);
#ifdef __CUDA_ARCH__
	//sample_importance(outputBuffer, *scene, params, coord, rng, importances, sums, impGrid);
#endif // __CUDA_ARCH__
}

__global__ static void impvis_kernel(pt::SilhouetteTargets::RenderBufferType<Device::CUDA> outputBuffer,
									 scene::SceneDescriptor<Device::CUDA>* scene,
									 const u32* seeds, Importances<Device::CUDA>** importances,
									 DeviceImportanceSums<Device::CUDA>* sums,
									 const float maxImportance) {
	Pixel coord{
		threadIdx.x + blockDim.x * blockIdx.x,
		threadIdx.y + blockDim.y * blockIdx.y
	};
	if(coord.x >= outputBuffer.get_width() || coord.y >= outputBuffer.get_height())
		return;

	const int pixel = coord.x + coord.y * outputBuffer.get_width();

	math::Rng rng(seeds[pixel]);
#ifdef __CUDA_ARCH__
	//sample_vis_importance(outputBuffer, *scene, coord, rng, importances, sums, maxImportance, impGrid);
#endif // __CUDA_ARCH__
}

namespace gpusil_details {


cudaError_t call_importance_kernel(const dim3& gridDims, const dim3& blockDims,
								   pt::SilhouetteTargets::RenderBufferType<Device::CUDA>&& outputBuffer,
								   scene::SceneDescriptor<Device::CUDA>* scene,
								   const u32* seeds, const SilhouetteParameters& params,
								   Importances<Device::CUDA>** importances,
								   DeviceImportanceSums<Device::CUDA>* sums) {
	silhouette_kernel<<<gridDims, blockDims>>>(std::move(outputBuffer), scene,
												  seeds, params, importances, sums);
	cudaDeviceSynchronize();
	return cudaGetLastError();
}

cudaError_t call_impvis_kernel(const dim3& gridDims, const dim3& blockDims,
							   pt::SilhouetteTargets::RenderBufferType<Device::CUDA>&& outputBuffer,
							   scene::SceneDescriptor<Device::CUDA>* scene,
							   const u32* seeds, Importances<Device::CUDA>** importances,
							   DeviceImportanceSums<Device::CUDA>* sums,
							   const float maxImportance) {
	impvis_kernel<<<gridDims, blockDims>>>(std::move(outputBuffer), scene,
										   seeds, importances, sums, maxImportance);
	cudaDeviceSynchronize();
	return cudaGetLastError();
}

} // namespace gpusil_details

}}}}} // namespace mufflon::renderer::decimaters::silhouette::pt
#endif // 0