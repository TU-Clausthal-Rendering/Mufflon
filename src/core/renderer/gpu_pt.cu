#include "hip/hip_runtime.h"
#include "gpu_pt.hpp"
#include "output_handler.hpp"
#include "core/cuda/error.hpp"
#include "core/math/rng.hpp"
#include "core/scene/lights/light_tree.hpp"
#include "core/scene/textures/interface.hpp"
#include "core/scene/accel_structs/intersection.hpp"
#include "core/scene/lights/light_tree_sampling.hpp"
#include "path_util.hpp"
#include "random_walk.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <random>

using namespace mufflon::scene::lights;

namespace mufflon { namespace renderer {

using PtPathVertex = PathVertex<u8, 4>;

__global__ static void sample(RenderBuffer<Device::CUDA> outputBuffer,
							  scene::SceneDescriptor<Device::CUDA>* scene,
							  const u32* seeds) {
	Pixel coord{
		threadIdx.x + blockDim.x * blockIdx.x,
		threadIdx.y + blockDim.y * blockIdx.y
	};
	if(coord.x >= outputBuffer.get_width() || coord.y >= outputBuffer.get_height())
		return;
	const u32 maxPathLength = 4u;

	int pixel = coord.x + coord.y * outputBuffer.get_width();

	math::Rng rng(seeds[pixel]);

	Throughput throughput{ ei::Vec3{1.0f}, 1.0f };
	u8 vertexBuffer[256]; // TODO: depends on materials::MAX_MATERIAL_PARAMETER_SIZE
	PtPathVertex* vertex = as<PtPathVertex>(vertexBuffer);
	// Create a start for the path
	int s = PtPathVertex::create_camera(vertex, vertex, scene->camera.get(), coord, rng.next());
	mAssertMsg(s < 256, "vertexBuffer overflow.");


#ifdef __CUDA_ARCH__
	int pathLen = 0;
	
	do {
		if(pathLen > 0 && pathLen + 1 <= maxPathLength) {
			// Call NEE member function for recursive vertices.
			// Do not connect to the camera, because this makes the renderer much more
			// complicated. Our decision: The PT should be as simple as possible!
			// What means more complicated?
			// A connnection to the camera results in a different pixel. In a multithreaded
			// environment this means that we need a write mutex for each pixel.
			// TODO: test/parametrize mulievent estimation (more indices in connect) and different guides.
			u64 neeSeed = rng.next();
			math::RndSet2 neeRnd = rng.next();
			auto nee = connect(scene->lightTree, 0, 1, neeSeed,
							   vertex->get_position(), scene->aabb,
							   neeRnd, scene::lights::guide_flux);
			auto value = vertex->evaluate(nee.direction, scene->media);
			mAssert(!isnan(value.value.x) && !isnan(value.value.y) && !isnan(value.value.z));
			Spectrum radiance = value.value * nee.diffIrradiance;
			if(any(greater(radiance, 0.0f)) && value.cosOut > 0.0f) {
				bool anyhit = scene::accel_struct::any_intersection_scene_lbvh<Device::CUDA>(
					*scene, { vertex->get_position() , nee.direction },
					vertex->get_primitive_id(), nee.dist);
				if(!anyhit) {
					AreaPdf hitPdf = value.pdfF.to_area_pdf(nee.cosOut, nee.distSq);
					float mis = 1.0f / (1.0f + hitPdf / nee.creationPdf);
					mAssert(!isnan(mis));
					outputBuffer.contribute(coord, throughput, { Spectrum{1.0f}, 1.0f },
											value.cosOut, radiance * mis);
				}
			}
		}

		// Walk
		scene::Point lastPosition = vertex->get_position();
		math::RndSet2_1 rnd{ rng.next(), rng.next() };
		math::DirectionSample lastDir;
		if(!walk(*scene, *vertex, rnd, -1.0f, false, throughput, vertex, lastDir)) {
			if(throughput.weight != Spectrum{ 0.f }) {
				// Missed scene - sample background
				auto background = evaluate_background(scene->lightTree.background, lastDir.direction);
				if(any(greater(background.value, 0.0f))) {
					float mis = 1.0f / (1.0f + background.pdfB / lastDir.pdf);
					background.value *= mis;
					outputBuffer.contribute(coord, throughput, background.value,
											ei::Vec3{ 0, 0, 0 }, ei::Vec3{ 0, 0, 0 },
											ei::Vec3{ 0, 0, 0 });
				}
			}
			break;
		}
		++pathLen;

		// Evaluate direct hit of area ligths
		if(pathLen <= maxPathLength) {
			Spectrum emission = vertex->get_emission();
			if(emission != 0.0f) {
				AreaPdf backwardPdf = connect_pdf(scene->lightTree, 0,
												  lastPosition, scene::lights::guide_flux);
				//float mis = 1.0f / (1.0f + backwardPdf / vertex->get_incident_pdf());
				float mis = 0.0f;
				outputBuffer.contribute(coord, throughput, emission, vertex->get_position(),
										vertex->get_normal(), vertex->get_albedo());
			}
		}
	} while(pathLen < maxPathLength);
#endif // __CUDA_ARCH__
}

void GpuPathTracer::iterate(Pixel imageDims,
							RenderBuffer<Device::CUDA> outputBuffer) const {
	
	std::unique_ptr<u32[]> rnds = std::make_unique<u32[]>(imageDims.x * imageDims.y);
	math::Xoroshiro128 rng{ static_cast<u32>(std::random_device()()) };
	for (int i = 0; i < imageDims.x*imageDims.y; ++i)
		rnds[i] = static_cast<u32>(rng.next());
	u32* devRnds = nullptr;
	cuda::check_error(hipMalloc(&devRnds, sizeof(u32) * imageDims.x * imageDims.y));
	cuda::check_error(hipMemcpy(devRnds, rnds.get(), sizeof(u32) * imageDims.x * imageDims.y,
		hipMemcpyDefault));

	// TODO: pass scene data to kernel!
	dim3 blockDims{ 16u, 16u, 1u };
	dim3 gridDims{
		1u + static_cast<u32>(imageDims.x - 1) / blockDims.x,
		1u + static_cast<u32>(imageDims.y - 1) / blockDims.y,
		1u
	};

	// TODO
	cuda::check_error(hipGetLastError());
	sample<<<gridDims, blockDims>>>(std::move(outputBuffer),
									m_scenePtr, devRnds);
	cuda::check_error(hipGetLastError());
	cuda::check_error(hipFree(devRnds));
}

}} // namespace mufflon::renderer