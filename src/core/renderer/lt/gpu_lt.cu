#include "hip/hip_runtime.h"
#include "lt_params.hpp"
#include "lt_common.hpp"
#include "core/cuda/error.hpp"
#include "core/math/rng.hpp"
#include "core/renderer/parameter.hpp"
#include <hip/hip_runtime.h>
#include <>

using namespace mufflon::scene::lights;

namespace mufflon { namespace renderer {

__global__ static void sample_lt(RenderBuffer<Device::CUDA> outputBuffer,
								 scene::SceneDescriptor<Device::CUDA>* scene,
								 math::Rng* rngs, LtParameters params) {
	Pixel coord{
		threadIdx.x + blockDim.x * blockIdx.x,
		threadIdx.y + blockDim.y * blockIdx.y
	};
	if(coord.x >= outputBuffer.get_width() || coord.y >= outputBuffer.get_height())
		return;

	const int idx = coord.x + coord.y * outputBuffer.get_width();

#ifdef __CUDA_ARCH__
	lt_sample(outputBuffer, *scene, params, idx, rngs[idx]);
#endif // __CUDA_ARCH__
}

namespace gpult_detail {

hipError_t call_kernel(RenderBuffer<Device::CUDA>&& outputBuffer,
						scene::SceneDescriptor<Device::CUDA>* scene,
						math::Rng* rngs, const LtParameters& params) {
	int minGridSize;
	int blockSize;
	cuda::check_error(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, sample_lt, 0));

	const dim3 blockDims{
		16u,
		static_cast<u32>(1 + (blockSize - 1) / 16),
		1u
	};
	const dim3 gridDims{
		1u + static_cast<u32>(outputBuffer.get_width() - 1) / blockDims.x,
		1u + static_cast<u32>(outputBuffer.get_height() - 1) / blockDims.y,
		1u
	};

	sample_lt<<<gridDims, blockDims>>>(std::move(outputBuffer), scene,
									   rngs, params);
	hipDeviceSynchronize();
	return hipGetLastError();
}

__global__ static void init_rng(u32 num, math::Rng* rngs) {
	u32 idx = threadIdx.x + blockDim.x * blockIdx.x;
	if(idx < num) {
		rngs[idx] = math::Rng{ idx };
	}
}
void init_rngs(u32 num, math::Rng* rngs) {
	dim3 blockDims { 256u, 1u, 1u };
	dim3 gridDims { (num + 255u) / 256u, 1u, 1u };
	init_rng<<<gridDims, blockDims>>>(num, rngs);
	//hipDeviceSynchronize();
	cuda::check_error(hipGetLastError());
}

} // namespace gpult_detail

}} // namespace mufflon::renderer
