#include "hip/hip_runtime.h"
#include "output_handler_variance.hpp"
#include "core/cuda/error.hpp"
#include <>

namespace mufflon { namespace renderer {

template < class T >
__global__ void update_variance_kernel(ConstRenderTargetBuffer<Device::CUDA, T> iterTarget,
									   RenderTargetBuffer<Device::CUDA, T> cumTarget,
									   RenderTargetBuffer<Device::CUDA, T> varTarget,
									   int numChannels,
									   int width, int height,
									   float iteration) {
#ifdef __CUDA_ARCH__
	int x = int(blockIdx.x * blockDim.x + threadIdx.x);
	int y = int(blockIdx.y * blockDim.y + threadIdx.y);
	if(x < width && y < height)
		output_handler_details::update_variance<T>(iterTarget, cumTarget, varTarget,
												   x, y, numChannels, width, iteration);
#endif
}

namespace output_handler_details {

template < class T >
void update_variance_cuda(ConstRenderTargetBuffer<Device::CUDA, T> iterTarget,
						  RenderTargetBuffer<Device::CUDA, T> cumTarget,
						  RenderTargetBuffer<Device::CUDA, T> varTarget,
						  int numChannels, int width, int height, int iteration) {
	dim3 dimBlock(16, 16);
	dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x,
		(height + dimBlock.y - 1) / dimBlock.y);
	update_variance_kernel<T><<<dimGrid, dimBlock>>>(
		iterTarget, cumTarget, varTarget, numChannels, width, height, float(iteration));
}

template void update_variance_cuda<float>(ConstRenderTargetBuffer<Device::CUDA, float>,
										  RenderTargetBuffer<Device::CUDA, float>,
										  RenderTargetBuffer<Device::CUDA, float>,
										  int, int, int, int);

__device__ u32 *s_nan_counter;

u32* get_cuda_nan_counter_ptr_and_set_zero() {
	constexpr u32 zero = 0;
	void* ptr = nullptr;
	cuda::check_error(::hipGetSymbolAddress(&ptr, HIP_SYMBOL(s_nan_counter)));
	cuda::check_error(::hipMemcpyToSymbolAsync(HIP_SYMBOL(s_nan_counter), &zero, sizeof(zero),
												0u, ::hipMemcpyHostToDevice));
	return reinterpret_cast<u32*>(ptr);
}

u32 get_cuda_nan_counter_value() {
	u32 counter = 0;
	cuda::check_error(::hipMemcpyFromSymbolAsync(&counter, HIP_SYMBOL(s_nan_counter), sizeof(counter),
												  0u, ::hipMemcpyDeviceToHost));
	return counter;
}

} // namespace output_handler_details

}} // namespace mufflon::renderer