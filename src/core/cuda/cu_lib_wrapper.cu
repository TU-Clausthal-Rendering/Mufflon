#include "cu_lib_wrapper.h"

#include <cub\cub.cuh>
//#include "Utilities\cudaHeaders.h"

#include <iostream>
#include <fstream>
#include <string>

namespace mufflon {namespace CuLib {


// In and out buffers may be swaped.
// Original data is not kept.
template <typename KeyT, typename ValueT> float DeviceSort(u32 numElements, 
	KeyT** keysIn, KeyT** keysOut,
	ValueT** valuesIn, ValueT** valuesOut)
{
	KeyT* tmpKeysOut;
	ValueT *tmpValuesOut;
	if (keysIn == keysOut) {
		hipMalloc((void **)&tmpKeysOut, numElements * sizeof(KeyT));
	}
	else {
		tmpKeysOut = *keysOut;
	}

	if (valuesIn == valuesOut) {
		hipMalloc((void **)&tmpValuesOut, numElements * sizeof(ValueT));
	}
	else {
		tmpValuesOut = *valuesOut;
	}

	hipcub::DoubleBuffer<KeyT> keysBuffer(*keysIn, tmpKeysOut);
	hipcub::DoubleBuffer<ValueT> valuesBuffer(*valuesIn, tmpValuesOut);

	// Check how much temporary memory will be required.
	void* tempStorage = nullptr;
	size_t storageSize = 0;
	hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, keysBuffer, valuesBuffer,
		numElements);

	// Allocate temporary memory.
	hipMalloc(&tempStorage, storageSize);

	float elapsedTime = 0.0f;
#ifdef MEASURE_EXECUTION_TIMES
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif

	// Sort
	hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, keysBuffer, valuesBuffer,
		numElements);

#ifdef MEASURE_EXECUTION_TIMES
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
#endif

	// Free temporary memory.
	hipFree(tempStorage);

	// Update in buffers.
	KeyT* current = keysBuffer.d_buffers[1 - keysBuffer.selector];
	if (keysIn != keysOut) {
		*keysIn = current;
	}
	else {
		hipFree(current);
	}
	ValueT* current2 = valuesBuffer.d_buffers[1 - valuesBuffer.selector];
	if (valuesIn != valuesOut) {
		*valuesIn = current2;
	}
	else {
		hipFree(current2);
	}

	// Update out buffers.
	current = keysBuffer.Current();
	*keysOut = current;
	current2 = valuesBuffer.Current();
	*valuesOut = current2;

	return elapsedTime;
}

#if 0
// In and out buffers may be swaped.
// Original data is not kept.
template <typename T> float DeviceSort(u32 numElements, T** keysIn, T** keysOut,
	u32** valuesIn, u32** valuesOut)
{
	T* tmpKeysOut;
	u32 *tmpValuesOut;
	if (keysIn == keysOut) {
		hipMalloc((void **)&tmpKeysOut, numElements * sizeof(T));
	}
	else {
		tmpKeysOut = *keysOut;
	}

	if (valuesIn == valuesOut) {
		hipMalloc((void **)&tmpValuesOut, numElements * sizeof(u32));
	}
	else {
		tmpValuesOut = *valuesOut;
	}

	hipcub::DoubleBuffer<T> keysBuffer(*keysIn, tmpKeysOut);
	hipcub::DoubleBuffer<u32> valuesBuffer(*valuesIn, tmpValuesOut);

	// Check how much temporary memory will be required.
	void* tempStorage = nullptr;
	size_t storageSize = 0;
	hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, keysBuffer, valuesBuffer,
		numElements);

	// Allocate temporary memory.
	hipMalloc(&tempStorage, storageSize);

	float elapsedTime = 0.0f;
#ifdef MEASURE_EXECUTION_TIMES
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif

	// Sort
	hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, keysBuffer, valuesBuffer,
		numElements);

#ifdef MEASURE_EXECUTION_TIMES
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
#endif

	// Free temporary memory.
	hipFree(tempStorage);

	// Update in buffers.
	T* current = keysBuffer.d_buffers[1 - keysBuffer.selector];
	if (keysIn != keysOut) {
		*keysIn = current;
	}
	else {
		hipFree(current);
	}
	u32* current2 = valuesBuffer.d_buffers[1 - valuesBuffer.selector];
	if (valuesIn != valuesOut) {
		*valuesIn = current2;
	}
	else {
		hipFree(current2);
	}

	// Update out buffers.
	current = keysBuffer.Current();
	*keysOut = current;
	current2 = valuesBuffer.Current();
	*valuesOut = current2;

	return elapsedTime;
}
#endif // 0


// In and out buffers may be swaped.
// Original data is not kept.
template <typename T> float DeviceSortDescending(u32 numElements, T** keysIn, T** keysOut,
	u32** valuesIn, u32** valuesOut)
{
	T* tmpKeysOut;
	u32 *tmpValuesOut;
	if (keysIn == keysOut) {
		hipMalloc((void **)&tmpKeysOut, numElements * sizeof(T));
	}
	else {
		tmpKeysOut = *keysOut;
	}

	if (valuesIn == valuesOut) {
		hipMalloc((void **)&tmpValuesOut, numElements * sizeof(u32));
	}
	else {
		tmpValuesOut = *valuesOut;
	}

	hipcub::DoubleBuffer<T> keysBuffer(*keysIn, tmpKeysOut);
	hipcub::DoubleBuffer<u32> valuesBuffer(*valuesIn, tmpValuesOut);

	// Check how much temporary memory will be required
	void* tempStorage = nullptr;
	size_t storageSize = 0;
	hipcub::DeviceRadixSort::SortPairsDescending(tempStorage, storageSize, keysBuffer, valuesBuffer,
		numElements);

	// Allocate temporary memory
	hipMalloc(&tempStorage, storageSize);

	float elapsedTime = 0.0f;
#ifdef MEASURE_EXECUTION_TIMES
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif

	// Sort.
	hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, keysBuffer, valuesBuffer,
		numElements);

#ifdef MEASURE_EXECUTION_TIMES
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
#endif

	// Free temporary memory
	hipFree(tempStorage);

	// Update in buffers
	T* current = keysBuffer.d_buffers[1 - keysBuffer.selector];
	if (keysIn != keysOut) {
		*keysIn = current;
	}
	else {
		hipFree(current);
	}
	u32* current2 = valuesBuffer.d_buffers[1 - valuesBuffer.selector];
	if (valuesIn != valuesOut) {
		*valuesIn = current2;
	}
	else {
		hipFree(current2);
	}

	// Update out buffers
	current = keysBuffer.Current();
	*keysOut = current;
	current2 = valuesBuffer.Current();
	*valuesOut = current2;

	return elapsedTime;
}


float DeviceSort(u32 numElements, u64 ** keysIn, u64 ** keysOut, u32 ** valuesIn, u32 ** valuesOut)
{
	return DeviceSort<u64, u32>(numElements, keysIn, keysOut, valuesIn, valuesOut);
}

float DeviceSort(u32 numElements, u64 ** keysIn, u64 ** keysOut, i32 ** valuesIn, i32 ** valuesOut)
{
	return DeviceSort<u64, i32>(numElements, keysIn, keysOut, valuesIn, valuesOut);
}

float DeviceSort(u32 numElements, u32 ** keysIn, u32 ** keysOut, i32 ** valuesIn, i32 ** valuesOut)
{
	return DeviceSort<u32, i32>(numElements, keysIn, keysOut, valuesIn, valuesOut);
}

float DeviceSort(u32 numElements, u32** keysIn, u32** keysOut,
	u32** valuesIn, u32** valuesOut)
{
	return DeviceSort<u32, u32>(numElements, keysIn, keysOut, valuesIn, valuesOut);
}

float DeviceSort(u32 numElements, float** keysIn, float** keysOut,
	u32** valuesIn, u32** valuesOut)
{
	return DeviceSort<float, u32>(numElements, keysIn, keysOut, valuesIn, valuesOut);
}

float DeviceSortDescending(u32 numElements, float** keysIn, float** keysOut,
	u32** valuesIn, u32** valuesOut)
{
	return DeviceSortDescending<float>(numElements, keysIn, keysOut, valuesIn, valuesOut);
}

//void DeviceSum(u32 numTriangles, int* in, int* out, size_t* tempMemorySize, void* tempMemory)
//{
//    hipcub::DeviceReduce::Sum(tempMemory, *tempMemorySize, in, out, numTriangles);
//}

template <class T> T DeviceSum(u32 numElements, T* elements)
{
	T* deviceElementsSum;
	hipMalloc(&deviceElementsSum, sizeof(T));

	// Calculate the required temporary memory size
	void* tempStorage = nullptr;
	size_t tempStorageSize = 0;
	hipcub::DeviceReduce::Sum(tempStorage, tempStorageSize, elements, deviceElementsSum,
		numElements);

	// Allocate temporary memory
	hipMalloc(&tempStorage, tempStorageSize);

	// Sum priorities
	hipcub::DeviceReduce::Sum(tempStorage, tempStorageSize, elements, deviceElementsSum,
		numElements);

	// Read priorities sum from device memory
	T elementsSum;
	hipMemcpy(&elementsSum, deviceElementsSum, sizeof(T), hipMemcpyDeviceToHost);

	// Free temporary memory
	hipFree(tempStorage);
	hipFree(deviceElementsSum);

	return elementsSum;
}

template <class T> T DeviceMax(u32 numElements, T* elements)
{
	T* deviceElementsSum;
	hipMalloc(&deviceElementsSum, sizeof(T));

	// Calculate the required temporary memory size
	void* tempStorage = nullptr;
	size_t tempStorageSize = 0;
	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, elements, deviceElementsSum,
		numElements);

	// Allocate temporary memory
	hipMalloc(&tempStorage, tempStorageSize);

	// Sum priorities
	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, elements, deviceElementsSum,
		numElements);

	// Read priorities sum from device memory
	T elementsSum;
	hipMemcpy(&elementsSum, deviceElementsSum, sizeof(T), hipMemcpyDeviceToHost);

	// Free temporary memory
	hipFree(tempStorage);
	hipFree(deviceElementsSum);

	return elementsSum;
}

int DeviceSum(u32 numElements, int* elements)
{
	return DeviceSum<int>(numElements, elements);
}

float DeviceSum(u32 numElements, float* elements)
{
	return DeviceSum<float>(numElements, elements);
}

int DeviceMax(u32 numElements, int* elements)
{
	return DeviceMax<int>(numElements, elements);
}

// ref. https://nvlabs.github.io/cub/structcub_1_1_device_scan.html#a83236fc272c0b573a2bb2c5b47e0867d
template <typename T> float DeviceExclusiveSum(int numItems, T** valuesIn, T** valuesOut)
{
	T* tmpValuesOut;
	if (valuesIn == valuesOut) {
		hipMalloc((void **)&tmpValuesOut, numItems * sizeof(T));
	}
	else {
		tmpValuesOut = *valuesOut;
	}

	// Check how much temporary memory will be required
	void* tempStorage = nullptr;
	size_t storageSize = 0;
	hipcub::DeviceScan::ExclusiveSum(tempStorage, storageSize, *valuesIn, tmpValuesOut, numItems);

	// Allocate temporary memory
	hipMalloc(&tempStorage, storageSize);

	float elapsedTime = 0.0f;
#ifdef MEASURE_EXECUTION_TIMES
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif

	// Scan
	hipcub::DeviceScan::ExclusiveSum(tempStorage, storageSize, *valuesIn, tmpValuesOut, numItems);

#ifdef MEASURE_EXECUTION_TIMES
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
#endif

	// Free temporary memory
	hipFree(tempStorage);

	if (valuesIn == valuesOut) {
		hipFree(*valuesIn);
		*valuesIn = tmpValuesOut;
	}

	return elapsedTime;
}

// ref. https://nvlabs.github.io/cub/structcub_1_1_device_scan.html#a83236fc272c0b573a2bb2c5b47e0867d
template <typename T> float DeviceInclusiveSum(int numItems, T** valuesIn, T** valuesOut)
{
	T* tmpValuesOut;
	if (valuesIn == valuesOut) {
		hipMalloc((void **)&tmpValuesOut, numItems * sizeof(T));
	}
	else {
		tmpValuesOut = *valuesOut;
	}

	// Check how much temporary memory will be required
	void* tempStorage = nullptr;
	size_t storageSize = 0;
	hipcub::DeviceScan::InclusiveSum(tempStorage, storageSize, *valuesIn, tmpValuesOut, numItems);

	// Allocate temporary memory
	hipMalloc(&tempStorage, storageSize);

	float elapsedTime = 0.0f;
#ifdef MEASURE_EXECUTION_TIMES
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif

	// Scan
	hipcub::DeviceScan::InclusiveSum(tempStorage, storageSize, *valuesIn, tmpValuesOut, numItems);

#ifdef MEASURE_EXECUTION_TIMES
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
#endif

	// Free temporary memory
	hipFree(tempStorage);

	if (valuesIn == valuesOut) {
		hipFree(*valuesIn);
		*valuesIn = tmpValuesOut;
	}

	return elapsedTime;
}


float DeviceExclusiveSum(int numElements, int** valuesIn, int** valuesOut)
{
	return DeviceExclusiveSum<int>(numElements, valuesIn, valuesOut);
}

float DeviceInclusiveSum(int numElements, int** valuesIn, int** valuesOut)
{
	return DeviceInclusiveSum<int>(numElements, valuesIn, valuesOut);
}

}}
